//
// Created by cheesema on 2019-07-09.
//

#include <hip/hip_runtime.h>
#include <>

#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>

#include "misc/CudaTools.cuh"
#include "misc/CudaMemory.cuh"
#include <chrono>
#include <cstdint>

#include "GPUAccess.hpp"

class GPUAccess::GPUAccessImpl{
public:
    ScopedCudaMemHandler<uint16_t*, JUST_ALLOC> y_vec;
    ScopedCudaMemHandler<uint64_t*, JUST_ALLOC> xz_end_vec;
    ScopedCudaMemHandler<uint64_t*, JUST_ALLOC> level_xz_vec;

    GPUAccessImpl()=default;
    ~GPUAccessImpl()=default;
};

GPUAccess::~GPUAccess() = default;
GPUAccess::GPUAccess(): data{new GPUAccessImpl}{

}
GPUAccess::GPUAccess(GPUAccess&&) = default;

void GPUAccess::init_y_vec(std::vector<uint16_t> &y_vec_) {
    data->y_vec.initialize(y_vec_.data(),y_vec_.size());
}

void GPUAccess::init_xz_end_vec(std::vector<uint64_t>& xz_end_vec){
    data->xz_end_vec.initialize(xz_end_vec.data(),xz_end_vec.size());
}
void GPUAccess::init_level_xz_vec(std::vector<uint64_t>& level_xz_vec){
    data->level_xz_vec.initialize(level_xz_vec.data(),level_xz_vec.size());
}


void GPUAccess::copy2Device(){
    data->y_vec.copyH2D();
    data->xz_end_vec.copyH2D();
    data->level_xz_vec.copyH2D();
}

void GPUAccess::copy2Device(const size_t numElements){
    data->y_vec.copyH2D(numElements);
    data->xz_end_vec.copyH2D();
    data->level_xz_vec.copyH2D();
}

void GPUAccess::copy2Host(){
    data->y_vec.copyD2H();
    data->xz_end_vec.copyD2H();
    data->level_xz_vec.copyD2H();
}

GPUAccessHelper::GPUAccessHelper(GPUAccess& gpuAccess_,LinearAccess& linearAccess_){
    gpuAccess = &gpuAccess_;
    linearAccess = &linearAccess_;
}

uint16_t* GPUAccessHelper::get_y_vec_ptr(){
    return gpuAccess->data->y_vec.get();
}

uint64_t* GPUAccessHelper::get_level_xz_vec_ptr(){
    return gpuAccess->data->level_xz_vec.get();
}

uint64_t* GPUAccessHelper::get_xz_end_vec_ptr(){
    return gpuAccess->data->xz_end_vec.get();
}



#include "data_structures/APR/particles/ParticleDataGpu.hpp"

template<typename DataType>
template<typename T>
class ParticleDataGpu<DataType>::ParticleDataGpuImpl {

public:
    ScopedCudaMemHandler<T *, H2D> part_data;

    ParticleDataGpuImpl() = default;

    ~ParticleDataGpuImpl() = default;

};

template<typename DataType>
ParticleDataGpu<DataType>::ParticleDataGpu(): data{new ParticleDataGpuImpl<DataType>}
{}

template<typename DataType>
ParticleDataGpu<DataType>::~ParticleDataGpu()
{}

template<typename DataType>
void ParticleDataGpu<DataType>::init(std::vector<DataType>& cpu_data){
    data->part_data.initialize(cpu_data.data(),cpu_data.size());
}

template<typename DataType>
DataType* ParticleDataGpu<DataType>::getGpuData(){
    return data->part_data.get();
}

template<typename DataType>
void ParticleDataGpu<DataType>::sendDataToGpu(){
    data->part_data.copyH2D();
}

template<typename DataType>
void ParticleDataGpu<DataType>::getDataFromGpu(){
    data->part_data.copyD2H();
}

template class ParticleDataGpu<uint16_t>;
template class ParticleDataGpu<uint8_t>;
template class ParticleDataGpu<float>;
template class ParticleDataGpu<double>;
template class ParticleDataGpu<int>;
template class ParticleDataGpu<uint64_t>;
