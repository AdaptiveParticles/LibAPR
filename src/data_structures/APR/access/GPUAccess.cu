//
// Created by cheesema on 2019-07-09.
//

#include <hip/hip_runtime.h>
#include <>

#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>

#include "misc/CudaTools.cuh"
#include "misc/CudaMemory.cuh"
#include <chrono>
#include <cstdint>

#include "GPUAccess.hpp"

class GPUAccess::GPUAccessImpl{
public:
    ScopedCudaMemHandler<uint16_t*, H2D> y_vec;
    ScopedCudaMemHandler<uint64_t*, H2D> xz_end_vec;
    ScopedCudaMemHandler<uint64_t*, H2D> level_xz_vec;

    GPUAccessImpl()=default;
    ~GPUAccessImpl()=default;
};

GPUAccess::~GPUAccess() = default;
GPUAccess::GPUAccess(): data{new GPUAccessImpl}{

}
GPUAccess::GPUAccess(GPUAccess&&) = default;

void GPUAccess::init_y_vec(std::vector<uint16_t> &y_vec_) {
    data->y_vec.initialize(y_vec_.data(),y_vec_.size());
}

void GPUAccess::init_xz_end_vec(std::vector<uint64_t>& xz_end_vec){
    data->xz_end_vec.initialize(xz_end_vec.data(),xz_end_vec.size());
}
void GPUAccess::init_level_xz_vec(std::vector<uint64_t>& level_xz_vec){
    data->level_xz_vec.initialize(level_xz_vec.data(),level_xz_vec.size());
}

void GPUAccess::copy2Device(){

}
void GPUAccess::copy2Host(){

}


