#include "PixelDataCuda.h"
#include <iostream>
#include <memory>

#include <hip/hip_runtime.h>
#include <>
//#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>

#include "misc/CudaTools.cuh"

#include "downsample.cuh"
#include <vector>

// explicit instantiation of handled types
template void downsampleMeanCuda(const PixelData<float>&, PixelData<float>&);
template void downsampleMaxCuda(const  PixelData<float>&, PixelData<float>&);

template <typename T, typename S>
void downsampleMeanCuda(const PixelData<T> &input, PixelData<S> &output) {
    ScopedCudaMemHandler<const PixelData<T>, H2D> in(input);
    ScopedCudaMemHandler<PixelData<S>, D2H> out(output);

    runDownsampleMean(in.get(), out.get(), input.x_num, input.y_num, input.z_num, 0);
};

template <typename T, typename S>
void downsampleMaxCuda(const PixelData<T> &input, PixelData<S> &output) {
    ScopedCudaMemHandler<const PixelData<T>, H2D> in(input);
    ScopedCudaMemHandler<PixelData<S>, D2H> out(output);

    runDownsampleMax(in.get(), out.get(), input.x_num, input.y_num, input.z_num, 0);
};
