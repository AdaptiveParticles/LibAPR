#include "PixelDataCuda.h"
#include <iostream>
#include <memory>

#include <hip/hip_runtime.h>
#include <>
//#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>

#include "misc/CudaTools.cuh"

#include "downsample.cuh"
#include "paddPixelData.cuh"


// explicit instantiation of handled types
template void downsampleMeanCuda(const PixelData<float>&, PixelData<float>&);
template void downsampleMaxCuda(const  PixelData<float>&, PixelData<float>&);
template void paddPixelsCuda(const PixelData<float> &input, PixelData<float> &output, const PixelDataDim &padSize);
template void unpaddPixelsCuda(const PixelData<float> &input, PixelData<float> &output, const PixelDataDim &padSize);

template <typename T, typename S>
void downsampleMeanCuda(const PixelData<T> &input, PixelData<S> &output) {
    ScopedCudaMemHandler<const PixelData<T>, H2D> in(input);
    ScopedCudaMemHandler<PixelData<S>, D2H> out(output);

    runDownsampleMean(in.get(), out.get(), input.x_num, input.y_num, input.z_num, 0);
};

template <typename T, typename S>
void downsampleMaxCuda(const PixelData<T> &input, PixelData<S> &output) {
    ScopedCudaMemHandler<const PixelData<T>, H2D> in(input);
    ScopedCudaMemHandler<PixelData<S>, D2H> out(output);

    runDownsampleMax(in.get(), out.get(), input.x_num, input.y_num, input.z_num, 0);
};

template <typename T>
void paddPixelsCuda(const PixelData<T> &input, PixelData<T> &output, const PixelDataDim &padSize) {
    ScopedCudaMemHandler<const PixelData<T>, H2D> inputData(input);
    ScopedCudaMemHandler<PixelData<T>, D2H> outputData(output);

    runPaddPixels(inputData.get(), outputData.get(), input.getDimension(), output.getDimension(), padSize, 0);
};

template <typename T>
void unpaddPixelsCuda(const PixelData<T> &input, PixelData<T> &output, const PixelDataDim &padSize) {
    ScopedCudaMemHandler<const PixelData<T>, H2D> inputData(input);
    ScopedCudaMemHandler<PixelData<T>, D2H> outputData(output);

    runUnpaddPixels(inputData.get(), outputData.get(), input.getDimension(), output.getDimension(), padSize, 0);
};
