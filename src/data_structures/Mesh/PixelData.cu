#include "PixelDataCuda.h"
#include <iostream>
#include <memory>

#include <hip/hip_runtime.h>
#include <>
#include <hip/device_functions.h>

#include "misc/CudaTools.hpp"

#include "downsample.cuh"

// explicit instantiation of handled types
template void downsampleMeanCuda(const PixelData<float>&, PixelData<float>&);
template void downsampleMaxCuda(const PixelData<float>&, PixelData<float>&);

template <typename T, typename S>
void downsampleMeanCuda(const PixelData<T> &input, PixelData<S> &output) {
    APRTimer timer(true);

    timer.start_timer("cuda: memory alloc + data transfer to device");

    size_t inputSize = input.mesh.size() * sizeof(T);
    T *cudaInput;
    hipMalloc(&cudaInput, inputSize);
    hipMemcpy(cudaInput, input.mesh.get(), inputSize, hipMemcpyHostToDevice);

    size_t outputSize = output.mesh.size() * sizeof(float);
    float *cudaOutput;
    hipMalloc(&cudaOutput, outputSize);
    hipMemcpy(cudaOutput, output.mesh.get(), outputSize, hipMemcpyHostToDevice);
    timer.stop_timer();

    timer.start_timer("cuda: calculations on device");
    dim3 threadsPerBlock(1, 64, 1);
    dim3 numBlocks(((input.x_num + threadsPerBlock.x - 1)/threadsPerBlock.x + 1) / 2,
                   (input.y_num + threadsPerBlock.y - 1)/threadsPerBlock.y,
                   ((input.z_num + threadsPerBlock.z - 1)/threadsPerBlock.z + 1) / 2);
    printCudaDims(threadsPerBlock, numBlocks);

    downsampleMeanKernel<<<numBlocks,threadsPerBlock>>>(cudaInput, cudaOutput, input.x_num, input.y_num, input.z_num);
    waitForCuda();
    timer.stop_timer();

    timer.start_timer("cuda: transfer data from device and freeing memory");
    hipFree(cudaInput);
    hipMemcpy((void*)output.mesh.get(), cudaOutput, outputSize, hipMemcpyDeviceToHost);
    hipFree(cudaOutput);
    timer.stop_timer();
};

template <typename T, typename S>
void downsampleMaxCuda(const PixelData<T> &input, PixelData<S> &output) {
    APRTimer timer(true);

    timer.start_timer("cuda: memory alloc + data transfer to device");

    size_t inputSize = input.mesh.size() * sizeof(T);
    T *cudaInput;
    hipMalloc(&cudaInput, inputSize);
    hipMemcpy(cudaInput, input.mesh.get(), inputSize, hipMemcpyHostToDevice);

    size_t outputSize = output.mesh.size() * sizeof(float);
    float *cudaOutput;
    hipMalloc(&cudaOutput, outputSize);
    hipMemcpy(cudaOutput, output.mesh.get(), outputSize, hipMemcpyHostToDevice);
    timer.stop_timer();

    timer.start_timer("cuda: calculations on device");
    dim3 threadsPerBlock(1, 64, 1);
    dim3 numBlocks(((input.x_num + threadsPerBlock.x - 1)/threadsPerBlock.x + 1) / 2,
                   (input.y_num + threadsPerBlock.y - 1)/threadsPerBlock.y,
                   ((input.z_num + threadsPerBlock.z - 1)/threadsPerBlock.z + 1) / 2);
    printCudaDims(threadsPerBlock, numBlocks);

    downsampleMaxKernel<<<numBlocks,threadsPerBlock>>>(cudaInput, cudaOutput, input.x_num, input.y_num, input.z_num);
    waitForCuda();
    timer.stop_timer();

    timer.start_timer("cuda: transfer data from device and freeing memory");
    hipFree(cudaInput);
    hipMemcpy((void*)output.mesh.get(), cudaOutput, outputSize, hipMemcpyDeviceToHost);
    hipFree(cudaOutput);
    timer.stop_timer();
};
