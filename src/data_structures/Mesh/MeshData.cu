#include "MeshDataCuda.h"
#include <iostream>
#include <memory>

#include <hip/hip_runtime.h>
#include <>
#include <hip/device_functions.h>

#include "downsample.cuh"

namespace {
    void waitForCuda() {
        hipDeviceSynchronize();
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) printf("Error: %s\n", hipGetErrorString(err));
    }

    void emptyCallForTemplateInstantiation() {
        MeshData<float> f = MeshData<float>(0, 0, 0);
        MeshData<uint16_t> u16 = MeshData<uint16_t>(0, 0, 0);
        MeshData<uint8_t> u8 = MeshData<uint8_t>(0, 0, 0);

        downsampleMeanCuda(f,  f);
        downsampleMeanCuda(u16,f);
        downsampleMeanCuda(u8, f);

        downsampleMaxCuda(f,  f);
        downsampleMaxCuda(u16,f);
        downsampleMaxCuda(u8, f);
    }

    void printCudaDims(const dim3 &threadsPerBlock, const dim3 &numBlocks) {
        std::cout << "Number of blocks  (x/y/z):  " << numBlocks.x << "/" << numBlocks.y << "/" << numBlocks.z << std::endl;
        std::cout << "Number of threads (x/y/z): " << threadsPerBlock.x << "/" << threadsPerBlock.y << "/" << threadsPerBlock.z << std::endl;
    }
}

template <typename T, typename S>
void downsampleMeanCuda(const MeshData<T> &input, MeshData<S> &output) {
    APRTimer timer(true);

    timer.start_timer("cuda: memory alloc + data transfer to device");

    size_t inputSize = input.mesh.size() * sizeof(T);
    T *cudaInput;
    hipMalloc(&cudaInput, inputSize);
    hipMemcpy(cudaInput, input.mesh.get(), inputSize, hipMemcpyHostToDevice);

    size_t outputSize = output.mesh.size() * sizeof(float);
    float *cudaOutput;
    hipMalloc(&cudaOutput, outputSize);
    hipMemcpy(cudaOutput, output.mesh.get(), outputSize, hipMemcpyHostToDevice);
    timer.stop_timer();

    timer.start_timer("cuda: calculations on device");
    dim3 threadsPerBlock(1, 64, 1);
    dim3 numBlocks(((input.x_num + threadsPerBlock.x - 1)/threadsPerBlock.x + 1) / 2,
                   (input.y_num + threadsPerBlock.y - 1)/threadsPerBlock.y,
                   ((input.z_num + threadsPerBlock.z - 1)/threadsPerBlock.z + 1) / 2);
    printCudaDims(threadsPerBlock, numBlocks);

    downsampleMeanKernel<<<numBlocks,threadsPerBlock>>>(cudaInput, cudaOutput, input.x_num, input.y_num, input.z_num);
    waitForCuda();
    timer.stop_timer();

    timer.start_timer("cuda: transfer data from device and freeing memory");
    hipFree(cudaInput);
    hipMemcpy((void*)output.mesh.get(), cudaOutput, outputSize, hipMemcpyDeviceToHost);
    hipFree(cudaOutput);
    timer.stop_timer();
};

template <typename T, typename S>
void downsampleMaxCuda(const MeshData<T> &input, MeshData<S> &output) {
    APRTimer timer(true);

    timer.start_timer("cuda: memory alloc + data transfer to device");

    size_t inputSize = input.mesh.size() * sizeof(T);
    T *cudaInput;
    hipMalloc(&cudaInput, inputSize);
    hipMemcpy(cudaInput, input.mesh.get(), inputSize, hipMemcpyHostToDevice);

    size_t outputSize = output.mesh.size() * sizeof(float);
    float *cudaOutput;
    hipMalloc(&cudaOutput, outputSize);
    hipMemcpy(cudaOutput, output.mesh.get(), outputSize, hipMemcpyHostToDevice);
    timer.stop_timer();

    timer.start_timer("cuda: calculations on device");
    dim3 threadsPerBlock(1, 64, 1);
    dim3 numBlocks(((input.x_num + threadsPerBlock.x - 1)/threadsPerBlock.x + 1) / 2,
                   (input.y_num + threadsPerBlock.y - 1)/threadsPerBlock.y,
                   ((input.z_num + threadsPerBlock.z - 1)/threadsPerBlock.z + 1) / 2);
    printCudaDims(threadsPerBlock, numBlocks);

    downsampleMaxKernel<<<numBlocks,threadsPerBlock>>>(cudaInput, cudaOutput, input.x_num, input.y_num, input.z_num);
    waitForCuda();
    timer.stop_timer();

    timer.start_timer("cuda: transfer data from device and freeing memory");
    hipFree(cudaInput);
    hipMemcpy((void*)output.mesh.get(), cudaOutput, outputSize, hipMemcpyDeviceToHost);
    hipFree(cudaOutput);
    timer.stop_timer();
};