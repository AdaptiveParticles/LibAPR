#include "hip/hip_runtime.h"
#include "LocalIntensityScaleCuda.h"

#include <iostream>
#include <memory>

#include <hip/hip_runtime.h>
#include <>
#include <math_functions.h>

namespace {
    void waitForCuda() {
        hipDeviceSynchronize();
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) printf("Error: %s\n", hipGetErrorString(err));
    }

    void emptyCallForTemplateInstantiation() {
        MeshData<float> f = MeshData<float>(0, 0, 0);
        MeshData<uint16_t> u16 = MeshData<uint16_t>(0, 0, 0);
        MeshData<uint8_t> u8 = MeshData<uint8_t>(0, 0, 0);

        calcMeanYdir(f, 0);
//        calcMeanYdir(u16, 0);
//        calcMeanYdir(u8, 0);
        calcMeanXdir(f, 0);
    }

    void printCudaDims(const dim3 &threadsPerBlock, const dim3 &numBlocks) {
        std::cout << "Number of blocks  (x/y/z):  " << numBlocks.x << "/" << numBlocks.y << "/" << numBlocks.z << std::endl;
        std::cout << "Number of threads (x/y/z): " << threadsPerBlock.x << "/" << threadsPerBlock.y << "/" << threadsPerBlock.z << std::endl;
    }
}
/**
 *
 * How it works along y-dir (let's suppose offset = 2 and number of workers = 8 for simplicity):
 *
 * image idx: 0 1 2 3 4 5 6 7 8 9 0 1 2 3 4 5 6 7 8 9 0 1 2
 *
 * loop #1
 * workersIdx 0 1 2 3 4 5 6 7
 * loop #2
 * workersIdx             6 7 0 1 2 3 4 5
 * loop #3
 * workersIdx                         4 5 6 7 0 1 2 3
 * ..............
 *
 * so #offset workers must wait in each loop to have next elements to sum
 *
 * @tparam T
 * @param image
 * @param offset
 * @param x_num
 * @param y_num
 * @param z_num
 */
template <typename T>
__global__ void meanYdir(T *image, int offset, size_t x_num, size_t y_num, size_t z_num) {
    // NOTE: Block size in x/z direction must be 1
    const size_t workersOffset = (blockIdx.z * x_num + blockIdx.x) * y_num;
    const int numOfWorkers = blockDim.y;
    const unsigned int active = __activemask();
    const int workerIdx = threadIdx.y;
    int workerOffset = workerIdx;

    int offsetInTheLoop = 0;
    T sum = 0;
    T v = 0;
    bool waitForNextLoop = false;
    int countNumOfSumElements = 1;
    while(workerOffset < y_num) {
        if (!waitForNextLoop) v = image[workersOffset + workerOffset];
        bool waitForNextValues = (workerIdx + offsetInTheLoop) % numOfWorkers >= (numOfWorkers - offset);
        for (int off = 1; off <= offset; ++off) {
            T prevElement = __shfl_sync(active, v, workerIdx + blockDim.y - off, blockDim.y);
            T nextElement = __shfl_sync(active, v, workerIdx + off, blockDim.y);
            // LHS boundary check + don't add previous values if they were added in a previous loop execution
            if (workerOffset >= off && !waitForNextLoop) {sum += prevElement; ++countNumOfSumElements;}
            // RHS boundary check + don't read next values since they are not read yet
            if (!waitForNextValues && workerOffset + off < y_num) {sum += nextElement; ++countNumOfSumElements;}
        }
        waitForNextLoop = waitForNextValues;
        if (!waitForNextLoop) {
            sum += v;
            image[workersOffset + workerOffset] = sum / countNumOfSumElements;

            // workere is done with current element - move to next one
            sum = 0;
            countNumOfSumElements = 1;
            workerOffset += numOfWorkers;
        }
        offsetInTheLoop += offset;
    }
}

template <typename T>
void calcMeanYdir(MeshData<T> &image, int offset) {
    APRTimer timer(true);

    timer.start_timer("cuda: memory alloc + data transfer to device");
    size_t imageSize = image.mesh.size() * sizeof(T);
    T *cudaImage;
    hipMalloc(&cudaImage, imageSize);
    hipMemcpy(cudaImage, image.mesh.get(), imageSize, hipMemcpyHostToDevice);
    timer.stop_timer();

    timer.start_timer("cuda: calculations on device");
    dim3 threadsPerBlock(1, 32, 1);
    dim3 numBlocks((image.x_num + threadsPerBlock.x - 1)/threadsPerBlock.x,
                   1,
                   (image.z_num + threadsPerBlock.z - 1)/threadsPerBlock.z);
    printCudaDims(threadsPerBlock, numBlocks);
    meanYdir<<<numBlocks,threadsPerBlock>>>(cudaImage, offset, image.x_num, image.y_num, image.z_num);
    waitForCuda();
    timer.stop_timer();

    timer.start_timer("cuda: transfer data from device and freeing memory");
    hipMemcpy((void*)image.mesh.get(), cudaImage, imageSize, hipMemcpyDeviceToHost);
    hipFree(cudaImage);
    timer.stop_timer();
}

template <typename T>
__global__ void meanXdir(T *image, int offset, size_t x_num, size_t y_num, size_t z_num) {
    const size_t workerOffset = blockIdx.y * blockDim.y + threadIdx.y + (blockIdx.z * blockDim.z + threadIdx.z) * y_num * x_num;
    const int workerYoffset = blockIdx.y * blockDim.y + threadIdx.y ;
    const int workerIdx = threadIdx.y;
    const int nextElementOffset = y_num;

    extern __shared__ float sharedMem[];
    float (*data)[32] = (float (*)[32])sharedMem;

    const int divisor = 2 * offset  + 1;
    int currElementOffset = 0;
    int saveElementOffset = 0;

    if (workerYoffset < y_num) {
        // clear shared mem
        for (int i = offset; i < divisor; ++i) data[i][workerIdx] = 0;

        // saturate cache
        float sum = 0;
        int count = 0;
        for (int i = 0; i < offset; ++i) {
            T v = image[workerOffset + currElementOffset];
            sum += v;
            data[i][workerIdx] = v;
            currElementOffset += nextElementOffset;
            ++count;
        }
        int bp = offset;

        // main loop
        for (int x = offset; x < x_num; ++x) {
            T v = image[workerOffset + currElementOffset];
            float sumT = sum;
            sum += v;
            sum -= data[bp][workerIdx];
            data[bp][workerIdx] = v;
            count = min(count + 1, divisor);
            image[workerOffset + saveElementOffset] = sum / count;
            bp = (bp + 1) % divisor;
            currElementOffset += nextElementOffset;
            saveElementOffset += nextElementOffset;
        }

        while (saveElementOffset < currElementOffset) {
            count = count - 1;
            sum -= data[bp][workerIdx];
            image[workerOffset + saveElementOffset] = sum / count;
            bp = (bp + 1) % divisor;
            saveElementOffset += nextElementOffset;
        }
    }

}

template <typename T>
void calcMeanXdir(MeshData<T> &image, int offset) {
    APRTimer timer(true);

    timer.start_timer("cuda: memory alloc + data transfer to device");
    size_t imageSize = image.mesh.size() * sizeof(T);
    T *cudaImage;
    hipMalloc(&cudaImage, imageSize);
    hipMemcpy(cudaImage, image.mesh.get(), imageSize, hipMemcpyHostToDevice);
    timer.stop_timer();

    timer.start_timer("cuda: calculations on device");
    dim3 threadsPerBlock(1, 32, 1);
    dim3 numBlocks(1,
                   (image.y_num + threadsPerBlock.y - 1)/threadsPerBlock.y,
                   (image.z_num + threadsPerBlock.z - 1)/threadsPerBlock.z);
    printCudaDims(threadsPerBlock, numBlocks);
    meanXdir<<<numBlocks,threadsPerBlock, (offset * 2 + 1) * sizeof(float) * 32>>>(cudaImage, offset, image.x_num, image.y_num, image.z_num);
    waitForCuda();
    timer.stop_timer();

    timer.start_timer("cuda: transfer data from device and freeing memory");
    hipMemcpy((void*)image.mesh.get(), cudaImage, imageSize, hipMemcpyDeviceToHost);
    hipFree(cudaImage);
    timer.stop_timer();
}