#include "hip/hip_runtime.h"
#include "LocalIntensityScaleCuda.h"

#include "LocalIntensityScale.hpp"

#include <iostream>
#include <memory>

#include <hip/hip_runtime.h>
#include <>
#include <math_functions.h>

#include "misc/CudaTools.hpp"


/**
 *
 * How it works along y-dir (let's suppose offset = 2 and number of workers = 8 for simplicity):
 *
 * image idx: 0 1 2 3 4 5 6 7 8 9 0 1 2 3 4 5 6 7 8 9 0 1 2
 *
 * loop #1
 * workersIdx 0 1 2 3 4 5 6 7
 * loop #2
 * workersIdx             6 7 0 1 2 3 4 5
 * loop #3
 * workersIdx                         4 5 6 7 0 1 2 3
 * ..............
 *
 * so #offset workers must wait in each loop to have next elements to sum
 *
 * @tparam T
 * @param image
 * @param offset
 * @param x_num
 * @param y_num
 * @param z_num
 */
template <typename T>
__global__ void meanYdir(T *image, int offset, size_t x_num, size_t y_num, size_t z_num) {
    // NOTE: Block size in x/z direction must be 1
    const size_t workersOffset = (blockIdx.z * x_num + blockIdx.x) * y_num;
    const int numOfWorkers = blockDim.y;
    const unsigned int active = __activemask();
    const int workerIdx = threadIdx.y;
    int workerOffset = workerIdx;

    int offsetInTheLoop = 0;
    T sum = 0;
    T v = 0;
    bool waitForNextLoop = false;
    int countNumOfSumElements = 1;
    while(workerOffset < y_num) {
        if (!waitForNextLoop) v = image[workersOffset + workerOffset];
        bool waitForNextValues = (workerIdx + offsetInTheLoop) % numOfWorkers >= (numOfWorkers - offset);
        for (int off = 1; off <= offset; ++off) {
            T prevElement = __shfl_sync(active, v, workerIdx + blockDim.y - off, blockDim.y);
            T nextElement = __shfl_sync(active, v, workerIdx + off, blockDim.y);
            // LHS boundary check + don't add previous values if they were added in a previous loop execution
            if (workerOffset >= off && !waitForNextLoop) {sum += prevElement; ++countNumOfSumElements;}
            // RHS boundary check + don't read next values since they are not read yet
            if (!waitForNextValues && workerOffset + off < y_num) {sum += nextElement; ++countNumOfSumElements;}
        }
        waitForNextLoop = waitForNextValues;
        if (!waitForNextLoop) {
            sum += v;
            image[workersOffset + workerOffset] = sum / countNumOfSumElements;

            // workere is done with current element - move to next one
            sum = 0;
            countNumOfSumElements = 1;
            workerOffset += numOfWorkers;
        }
        offsetInTheLoop += offset;
    }
}

constexpr int NumberOfWorkers = 32; // Cannot be greater than 32 since there is no inter-warp communication implemented.

/**
 * Filter in X-dir moves circular buffer along direction adding to sum of elements newly read element and removing last one.
 * For instance (filter len = 5)
 *
 * idx:               0 1 2 3 4 5 6 7 8 9
 * image elements:    1 2 2 4 5 3 2 1 3 4
 *
 * buffer:                2 3 4 5 2                        current sum = 16  element @idx=4 will be updated to 16/5
 *
 * next step
 * buffer:                  3 4 5 2 1                      sum = sum - 2 + 1 = 15  element @idx=5 = 15 / 5
 *
 * In general circular buffer is kept to speedup operations and to not reach to global memory more than once for
 * read/write operations for given element.
 */
template <typename T>
__global__ void meanXdir(T *image, int offset, size_t x_num, size_t y_num, size_t z_num) {
    const size_t workerOffset = blockIdx.y * blockDim.y + threadIdx.y + (blockIdx.z * blockDim.z + threadIdx.z) * y_num * x_num;
    const int workerYoffset = blockIdx.y * blockDim.y + threadIdx.y ;
    const int workerIdx = threadIdx.y;
    const int nextElementOffset = y_num;

    extern __shared__ float sharedMem[];
    float (*data)[NumberOfWorkers] = (float (*)[NumberOfWorkers])sharedMem;

    const int divisor = 2 * offset  + 1;
    int currElementOffset = 0;
    int saveElementOffset = 0;

    if (workerYoffset < y_num) {
        // clear shared mem
        for (int i = offset; i < divisor; ++i) data[i][workerIdx] = 0;

        // saturate cache with #offset elements since it will allow to calculate first element value on LHS
        float sum = 0;
        int count = 0;
        while (count < offset) {
            T v = image[workerOffset + currElementOffset];
            sum += v;
            data[count][workerIdx] = v;
            currElementOffset += nextElementOffset;
            ++count;
        }

        // Pointer in circular buffer
        int beginPtr = offset;

        // main loop going through all elements in range [0, x_num-offset)
        for (int x = 0; x < x_num - offset; ++x) {
            // Read new element
            T v = image[workerOffset + currElementOffset];

            // Update sum to cover [-offset, offset] of currently processed element
            sum += v;
            sum -= data[beginPtr][workerIdx];

            // Save and move pointer
            data[beginPtr][workerIdx] = v;
            beginPtr = (beginPtr + 1) % divisor;

            // Update count and save currently processed element
            count = min(count + 1, divisor);
            image[workerOffset + saveElementOffset] = sum / count;

            // Move to next elements
            currElementOffset += nextElementOffset;
            saveElementOffset += nextElementOffset;
        }

        // Handle last #offset elements on RHS
        while (saveElementOffset < currElementOffset) {
            count = count - 1;
            sum -= data[beginPtr][workerIdx];
            image[workerOffset + saveElementOffset] = sum / count;
            beginPtr = (beginPtr + 1) % divisor;
            saveElementOffset += nextElementOffset;
        }
    }
}

/**
 * Filter in Z-dir moves circular buffer along direction adding to sum of elements newly read element and removing last one.
 * For instance (filter len = 5)
 *
 * idx:               0 1 2 3 4 5 6 7 8 9
 * image elements:    1 2 2 4 5 3 2 1 3 4
 *
 * buffer:                2 3 4 5 2                        current sum = 16  element @idx=4 will be updated to 16/5
 *
 * next step
 * buffer:                  3 4 5 2 1                      sum = sum - 2 + 1 = 15  element @idx=5 = 15 / 5
 *
 * In general circular buffer is kept to speedup operations and to not reach to global memory more than once for
 * read/write operations for given element.
 */
template <typename T>
__global__ void meanZdir(T *image, int offset, size_t x_num, size_t y_num, size_t z_num) {
    const size_t workerOffset = blockIdx.y * blockDim.y + threadIdx.y + (blockIdx.z * blockDim.z + threadIdx.z) * y_num; // *.z is 'x'
    const int workerYoffset = blockIdx.y * blockDim.y + threadIdx.y ;
    const int workerIdx = threadIdx.y;
    const int nextElementOffset = x_num * y_num;

    extern __shared__ float sharedMem[];
    float (*data)[NumberOfWorkers] = (float (*)[NumberOfWorkers])sharedMem;

    const int divisor = 2 * offset  + 1;
    int currElementOffset = 0;
    int saveElementOffset = 0;

    if (workerYoffset < y_num) {
        // clear shared mem
        for (int i = offset; i < divisor; ++i) data[i][workerIdx] = 0;

        // saturate cache with #offset elements since it will allow to calculate first element value on LHS
        float sum = 0;
        int count = 0;
        while (count < offset) {
            T v = image[workerOffset + currElementOffset];
            sum += v;
            data[count][workerIdx] = v;
            currElementOffset += nextElementOffset;
            ++count;
        }

        // Pointer in circular buffer
        int beginPtr = offset;

        // main loop going through all elements in range [0, x_num-offset)
        for (int z = 0; z < z_num - offset; ++z) {
            // Read new element
            T v = image[workerOffset + currElementOffset];

            // Update sum to cover [-offset, offset] of currently processed element
            sum += v;
            sum -= data[beginPtr][workerIdx];

            // Save and move pointer
            data[beginPtr][workerIdx] = v;
            beginPtr = (beginPtr + 1) % divisor;

            // Update count and save currently processed element
            count = min(count + 1, divisor);
            image[workerOffset + saveElementOffset] = sum / count;

            // Move to next elements
            currElementOffset += nextElementOffset;
            saveElementOffset += nextElementOffset;
        }

        // Handle last #offset elements on RHS
        while (saveElementOffset < currElementOffset) {
            count = count - 1;
            sum -= data[beginPtr][workerIdx];
            image[workerOffset + saveElementOffset] = sum / count;
            beginPtr = (beginPtr + 1) % divisor;
            saveElementOffset += nextElementOffset;
        }
    }
}

template <typename T, typename S>
void localIntensityScaleCUDA(T *cudaImage, const PixelData<S> &image, int offsetX, int offsetY, int offsetZ, TypeOfMeanFlags flags) {
    APRTimer timer(true);


    if (flags & MEAN_Y_DIR) {
        timer.start_timer("GpuDeviceTimeYdirLIS");
        dim3 threadsPerBlock(1, NumberOfWorkers, 1);
        dim3 numBlocks((image.x_num + threadsPerBlock.x - 1)/threadsPerBlock.x,
                       1,
                       (image.z_num + threadsPerBlock.z - 1)/threadsPerBlock.z);
        printCudaDims(threadsPerBlock, numBlocks);
        meanYdir<<<numBlocks,threadsPerBlock>>>(cudaImage, offsetY, image.x_num, image.y_num, image.z_num);
//        waitForCuda();
        timer.stop_timer();
    }

    if (flags & MEAN_X_DIR) {
        // Shared memory size  - it is able to keep filter len elements for each worker.
        const int sharedMemorySize = (offsetX * 2 + 1) * sizeof(float) * NumberOfWorkers;
        timer.start_timer("GpuDeviceTimeXdirLIS");
        dim3 threadsPerBlock(1, NumberOfWorkers, 1);
        dim3 numBlocks(1,
                       (image.y_num + threadsPerBlock.y - 1) / threadsPerBlock.y,
                       (image.z_num + threadsPerBlock.z - 1) / threadsPerBlock.z);
        printCudaDims(threadsPerBlock, numBlocks);
        meanXdir <<< numBlocks, threadsPerBlock, sharedMemorySize >>> (cudaImage, offsetX, image.x_num, image.y_num, image.z_num);
//        waitForCuda();
        timer.stop_timer();
    }
    if (flags & MEAN_Z_DIR) {
        // Shared memory size  - it is able to keep filter len elements for each worker.
        const int sharedMemorySize = (offsetZ * 2 + 1) * sizeof(float) * NumberOfWorkers;
        timer.start_timer("GpuDeviceTimeZdirLIS");
        dim3 threadsPerBlock(1, NumberOfWorkers, 1);
        dim3 numBlocks(1,
                       (image.y_num + threadsPerBlock.y - 1) / threadsPerBlock.y,
                       (image.x_num + threadsPerBlock.x - 1) / threadsPerBlock.x); // intentionally here for better memory readings
        printCudaDims(threadsPerBlock, numBlocks);
        meanZdir <<< numBlocks, threadsPerBlock, sharedMemorySize >>> (cudaImage, offsetZ, image.x_num, image.y_num, image.z_num);
//        waitForCuda();
        timer.stop_timer();
    }
}

template <typename T>
void calcMean(PixelData<T> &image, int offset, TypeOfMeanFlags flags) {
    APRTimer timer(true);

    timer.start_timer("GpuMemTransferHostToDevice");
    size_t imageSize = image.mesh.size() * sizeof(T);
    T *cudaImage;
    hipMalloc(&cudaImage, imageSize);
    hipMemcpy(cudaImage, image.mesh.get(), imageSize, hipMemcpyHostToDevice);
    timer.stop_timer();

    // --------- CUDA ----------------
    timer.start_timer("GpuDeviceTimeFull");
    localIntensityScaleCUDA(cudaImage, image, offset, offset, offset, flags);
    timer.stop_timer();

    timer.start_timer("cuda: transfer data from device and freeing memory");
    hipMemcpy((void*)image.mesh.get(), cudaImage, imageSize, hipMemcpyDeviceToHost);
    hipFree(cudaImage);
    timer.stop_timer();
}

template <typename T>
__global__ void copy1dKernel(const T *input, T *output, size_t len) {
    size_t idx = (size_t)blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < len) {
        output[idx] = input[idx];
    }
}

template <typename T>
void copy1d(const T *input, T *output, size_t len) {
    dim3 threadsPerBlock(64);
    dim3 numBlocks((len + threadsPerBlock.x - 1) / threadsPerBlock.x);
    copy1dKernel <<< numBlocks, threadsPerBlock >>> (input, output, len);
}

template<typename T>
__global__ void absDiff1dKernel(T *data, const T *reference, size_t len) {
    size_t idx = (size_t)blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < len) {
        data[idx] = abs(data[idx] - reference[idx]);
    }
}

template <typename T>
void absDiff1d(T *data, const T *reference, size_t len) {
    dim3 threadsPerBlock(64);
    dim3 numBlocks((len + threadsPerBlock.x - 1) / threadsPerBlock.x);
    absDiff1dKernel <<< numBlocks, threadsPerBlock >>> (data, reference, len);
}

template<typename T>
__global__ void rescaleKernel(T *data, size_t len, float varRescale, float sigmaThreshold, float sigmaThresholdMax) {
    const float max_th = 60000.0;
    size_t idx = (size_t)blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < len) {
        float rescaled = varRescale * data[idx];
        if (rescaled < sigmaThreshold) {
            rescaled = (rescaled < sigmaThresholdMax) ? max_th : sigmaThreshold;
        }
        data[idx] = rescaled;
    }
}

template <typename T>
void rescale(T *data, size_t len, float varRescale, float sigma, float sigmaMax) {
    dim3 threadsPerBlock(64);
    dim3 numBlocks((len + threadsPerBlock.x - 1) / threadsPerBlock.x);
    rescaleKernel <<< numBlocks, threadsPerBlock >>> (data, len, varRescale, sigma, sigmaMax);
}

template <typename T, typename S>
void localIntensityScaleCuda(const PixelData<T> &image, const APRParameters &par, S *cudaImage, S *cudaTemp) {
    CudaTimer timer(true, "localIntensityScaleCuda");

    float var_rescale;
    std::vector<int> var_win;
    LocalIntensityScale().get_window(var_rescale,var_win,par);
    size_t win_y = var_win[0];
    size_t win_x = var_win[1];
    size_t win_z = var_win[2];
    size_t win_y2 = var_win[3];
    size_t win_x2 = var_win[4];
    size_t win_z2 = var_win[5];
    std::cout << "GPU WINDOWS: " << win_y << " " << win_x << " " << win_z << " " << win_y2 << " " << win_x2 << " " << win_z2 << std::endl;
    // --------- CUDA ----------------
    timer.start_timer("copy_intensities_from_bsplines");
    copy1d(cudaImage, cudaTemp, image.mesh.size());
    timer.stop_timer();


    localIntensityScaleCUDA(cudaImage, image, win_x, win_y, win_z, MEAN_ALL_DIR);

    timer.start_timer("second_pass_and_rescale");
    absDiff1d(cudaImage, cudaTemp, image.mesh.size());
    localIntensityScaleCUDA(cudaImage, image, win_x2, win_y2, win_z2, MEAN_ALL_DIR);
    rescale(cudaImage, image.mesh.size(), var_rescale, par.sigma_th, par.sigma_th_max);
    timer.stop_timer();
}

template <typename T>
void getLocalIntensityScale(PixelData<T> &image, PixelData<T> &temp, const APRParameters &par) {
    APRTimer timer(true), timerFullPipelilne(true);

    timer.start_timer("GpuMemTransferHostToDevice");
    size_t imageSize = image.mesh.size() * sizeof(T);
    T *cudaImage;
    hipMalloc(&cudaImage, imageSize);
    hipMemcpy(cudaImage, image.mesh.get(), imageSize, hipMemcpyHostToDevice);
    T *cudaTemp;
    hipMalloc(&cudaTemp, imageSize);
    timer.stop_timer();

    timerFullPipelilne.start_timer("GpuDeviceTimeFull");
    localIntensityScaleCuda(image, par, cudaImage, cudaTemp);
    timerFullPipelilne.stop_timer();

    timer.start_timer("GpuMemTransferDeviceToHost");
    getDataFromKernel(image, imageSize, cudaImage);
    getDataFromKernel(temp, imageSize, cudaTemp);
    timer.stop_timer();
}

// explicit instantiation of handled types
template void calcMean(PixelData<float>&, int, TypeOfMeanFlags);
template void calcMean(PixelData<uint16_t>&, int, TypeOfMeanFlags);
template void calcMean(PixelData<uint8_t>&, int, TypeOfMeanFlags);

template void getLocalIntensityScale(PixelData<float>&, PixelData<float>&, const APRParameters&);
