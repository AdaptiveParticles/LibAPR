#include "hip/hip_runtime.h"
#include "LocalIntensityScaleCuda.h"

#include <iostream>
#include <memory>

#include <hip/hip_runtime.h>
#include <>

namespace {
    void waitForCuda() {
        hipDeviceSynchronize();
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) printf("Error: %s\n", hipGetErrorString(err));
    }

    void emptyCallForTemplateInstantiation() {
        MeshData<float> f = MeshData<float>(0, 0, 0);
        MeshData<uint16_t> u16 = MeshData<uint16_t>(0, 0, 0);
        MeshData<uint8_t> u8 = MeshData<uint8_t>(0, 0, 0);

        calcMeanYdir(f, 0);
//        calcMeanYdir(u16, 0);
//        calcMeanYdir(u8, 0);
    }

    void printCudaDims(const dim3 &threadsPerBlock, const dim3 &numBlocks) {
        std::cout << "Number of blocks  (x/y/z):  " << numBlocks.x << "/" << numBlocks.y << "/" << numBlocks.z << std::endl;
        std::cout << "Number of threads (x/y/z): " << threadsPerBlock.x << "/" << threadsPerBlock.y << "/" << threadsPerBlock.z << std::endl;
    }
}

template <typename T>
__global__ void meanYdir(T *image, int offset, size_t x_num, size_t y_num, size_t z_num) {
    // NOTE: Block size in x/z direction must be 1
    const size_t workersOffset = (blockIdx.z * x_num + blockIdx.x) * y_num;
    const int numOfWorkers = blockDim.y;
    const unsigned int active = __activemask();
    const int workerIdx = threadIdx.y;
    int workerOffset = workerIdx;

    int offsetInTheLoop = 0;
    T sum = 0;
    T v = 0;
    bool waitForNextLoop = false;
    int countNumOfSumElements = 1;
    while(workerOffset < y_num) {
        if (!waitForNextLoop) v = image[workersOffset + workerOffset];
        bool waitForNextValues = (workerIdx + offsetInTheLoop) % numOfWorkers >= (numOfWorkers - offset);
        for (int off = 1; off <= offset; ++off) {
            T prevElement = __shfl_sync(active, v, workerIdx + blockDim.y - off, blockDim.y);
            T nextElement = __shfl_sync(active, v, workerIdx + off, blockDim.y);
            // LHS boundary check + don't add previous values if they were added in a previous loop execution
            if (workerOffset >= off && !waitForNextLoop) {sum += prevElement; ++countNumOfSumElements;}
            // RHS boundary check + don't read next values since they are not read yet
            if (!waitForNextValues && workerOffset + off < y_num) {sum += nextElement; ++countNumOfSumElements;}
        }
        waitForNextLoop = waitForNextValues;
        if (!waitForNextLoop) {
            sum += v;
            image[workersOffset + workerOffset] = sum / countNumOfSumElements;

            // workere is done with current element - move to next one
            sum = 0;
            countNumOfSumElements = 1;
            workerOffset += numOfWorkers;
        }
        offsetInTheLoop += offset;
    }
}

template <typename T>
void calcMeanYdir(MeshData<T> &image, int offset) {
    APRTimer timer(true);

    timer.start_timer("cuda: memory alloc + data transfer to device");
    size_t imageSize = image.mesh.size() * sizeof(T);
    T *cudaImage;
    hipMalloc(&cudaImage, imageSize);
    hipMemcpy(cudaImage, image.mesh.get(), imageSize, hipMemcpyHostToDevice);
    timer.stop_timer();

    timer.start_timer("cuda: calculations on device");
    dim3 threadsPerBlock(1, 32, 1);
    dim3 numBlocks((image.x_num + threadsPerBlock.x - 1)/threadsPerBlock.x,
                   1,
                   (image.z_num + threadsPerBlock.z - 1)/threadsPerBlock.z);
    printCudaDims(threadsPerBlock, numBlocks);
    meanYdir<<<numBlocks,threadsPerBlock>>>(cudaImage, offset, image.x_num, image.y_num, image.z_num);
    waitForCuda();
    timer.stop_timer();

    timer.start_timer("cuda: transfer data from device and freeing memory");
    hipMemcpy((void*)image.mesh.get(), cudaImage, imageSize, hipMemcpyDeviceToHost);
    hipFree(cudaImage);
    timer.stop_timer();
}
