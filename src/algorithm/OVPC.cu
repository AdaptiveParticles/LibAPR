#include "hip/hip_runtime.h"
#include "PullingSchemeCuda.hpp"

#include <hip/hip_runtime.h>
#include <>
#include <hip/device_functions.h>

#include "misc/CudaTools.cuh"
#include "data_structures/Mesh/downsample.cuh"

namespace {
    using ElementType = uint8_t;
    static constexpr int BIT_SHIFT = 6;
    static constexpr ElementType OVPC_SEED = 1;
    static constexpr ElementType OVPC_BOUNDARY = 2;
    static constexpr ElementType OVPC_FILLER = 3;

    static constexpr ElementType  SEED_MASK = OVPC_SEED << BIT_SHIFT;
    static constexpr ElementType  BOUNDARY_MASK = OVPC_BOUNDARY << BIT_SHIFT;
    static constexpr ElementType  FILLER_MASK = OVPC_FILLER << BIT_SHIFT;
    static constexpr ElementType  MASK = 0x03 << BIT_SHIFT;
}

template <typename T, typename S>
__global__ void copy1D(const T *input, S *output, size_t length) {
    size_t idx = (size_t)blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < length) {
        output[idx] = input[idx];
    }
}

template <typename T, typename S>
void runCopy1D(T *inputData, S *outputData, size_t lenght, hipStream_t aStream) {
    dim3 threadsPerBlock(128);
    dim3 numBlocks((lenght + threadsPerBlock.x - 1)/threadsPerBlock.x);
    copy1D<<<numBlocks,threadsPerBlock, 0, aStream>>>(inputData, outputData, lenght);
};


template <typename T>
__global__ void oneLevel(T *data, size_t xLen, size_t yLen, size_t zLen, int level) {
    const int xi = (blockIdx.x * blockDim.x) + threadIdx.x;
    const int yi = (blockIdx.y * blockDim.y) + threadIdx.y;
    const int zi = (blockIdx.z * blockDim.z) + threadIdx.z;
    if (xi >= xLen || yi >= yLen || zi >= zLen) return;

    int xmin = xi > 0 ? xi - 1 : 0;
    int xmax = xi < xLen - 1 ? xi + 1 : xLen - 1;
    int ymin = yi > 0 ? yi - 1 : 0;
    int ymax = yi < yLen - 1 ? yi + 1 : yLen - 1;
    int zmin = zi > 0 ? zi - 1 : 0;
    int zmax = zi < zLen - 1 ? zi + 1 : zLen - 1;

    bool ok = true;
    bool neig = false;
    for (int z = zmin; z <= zmax; ++z) {
        for (int x = xmin; x <= xmax; ++x) {
            for (int y = ymin; y <= ymax; ++y) {
                const size_t idx = z * xLen * yLen + x * yLen + y;
                T currentLevel = ~MASK & data[idx];
                if (currentLevel > level) { ok = false; break; }
                else if (currentLevel == level) neig = true;
            }
        }
    }
    if (ok) {
        const size_t idx = zi * xLen * yLen + xi * yLen + yi;
        T status = data[idx];
        if (status == level) data[idx] |= SEED_MASK;
        else if (neig) data[idx] |= BOUNDARY_MASK;
        else data[idx] |= FILLER_MASK;
    }
}

template <typename T>
void runOneLevel(T *data, size_t xLen, size_t yLen, size_t zLen, int level, hipStream_t aStream) {
    dim3 threadsPerBlock(1, 128, 1);
    dim3 numBlocks((xLen + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (yLen + threadsPerBlock.y - 1) / threadsPerBlock.y,
                   (zLen + threadsPerBlock.z - 1) / threadsPerBlock.z);
//    dim3 numBlocks((xLen * yLen * zLen + threadsPerBlock.x - 1)/threadsPerBlock.x);
    oneLevel<<<numBlocks,threadsPerBlock, 0, aStream>>>(data, xLen, yLen, zLen, level);
};

template <typename T>
__global__ void secondPhase(T *data, T *child, size_t xLen, size_t yLen, size_t zLen, size_t xLenc, size_t yLenc, size_t zLenc, bool isLevelMax) {
    const int xi = (blockIdx.x * blockDim.x) + threadIdx.x;
    const int yi = (blockIdx.y * blockDim.y) + threadIdx.y;
    const int zi = (blockIdx.z * blockDim.z) + threadIdx.z;
    if (xi >= xLen || yi >= yLen || zi >= zLen) return;

    int xmin = 2 * xi;
    int xmax = 2 * xi + 1; xmax = xmax >= xLenc ? xLenc - 1 : xmax;
    int ymin = 2 * yi;
    int ymax = 2 * yi + 1; ymax = ymax >= yLenc ? yLenc - 1 : ymax;
    int zmin = 2 * zi;
    int zmax = 2 * zi + 1; zmax = zmax >= zLenc ? zLenc - 1 : zmax;

    uint8_t status = data[zi * xLen * yLen + xi * yLen + yi];

    for (int z = zmin; z <= zmax; ++z) {
        for (int x = xmin; x <= xmax; ++x) {
            for (int y = ymin; y <= ymax; ++y) {
                size_t children_index = z * xLenc * yLenc + x * yLenc + y;
                ElementType  v = child[children_index];
                child[children_index] = status >= (OVPC_SEED << BIT_SHIFT) ? 0 : v >> BIT_SHIFT;
            }
        }
    }
    if (isLevelMax) data[zi * xLen * yLen + xi * yLen + yi] = status >> BIT_SHIFT;
}

template <typename T>
void runSecondPhase(T *data, T *child, size_t xLen, size_t yLen, size_t zLen, size_t xLenc, size_t yLenc, size_t zLenc, bool isLevelMax, hipStream_t aStream) {
    dim3 threadsPerBlock(1, 128, 1);
    dim3 numBlocks((xLen + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (yLen + threadsPerBlock.y - 1) / threadsPerBlock.y,
                   (zLen + threadsPerBlock.z - 1) / threadsPerBlock.z);
    secondPhase<<<numBlocks,threadsPerBlock, 0, aStream>>>(data, child, xLen, yLen, zLen, xLenc, yLenc, zLenc, isLevelMax);
};

// explicit instantiation of handled types
template void computeOVPC(const PixelData<float>&, PixelData<TreeElementType>&, int, int);

template <typename T, typename S>
void computeOVPC(const PixelData<T> &input, PixelData<S> &output, int levelMin, int levelMax) {
    ScopedCudaMemHandler<const PixelData<T>, H2D> in(input);
    ScopedCudaMemHandler<PixelData<S>, D2H> mem(output);

    // TODO: This is not needed later - just for having clear debug
    //hipMemset(mem.get(), 0, mem.getNumOfBytes());

    // =============== Create pyramid
    std::vector<S*> levels(levelMax + 1, nullptr);
    std::vector<size_t> xSize(levelMax + 1);
    std::vector<size_t> ySize(levelMax + 1);
    std::vector<size_t> zSize(levelMax + 1);

    int xDS = input.x_num;
    int yDS = input.y_num;
    int zDS = input.z_num;

    size_t offset = 0;
    for (int l = levelMax; l >= levelMin; --l) {
        levels[l] = reinterpret_cast<TreeElementType *>(mem.get()) + offset;
        xSize[l] = xDS;
        ySize[l] = yDS;
        zSize[l] = zDS;

        offset += xDS * yDS * zDS * sizeof(TreeElementType);
        // round up to 16-bytes
        const size_t alignemet = 16;
        offset = ((offset + alignemet - 1) / alignemet ) * alignemet;

        xDS = ceil(xDS/2.0);
        yDS = ceil(yDS/2.0);
        zDS = ceil(zDS/2.0);
    }

    runCopy1D(in.get(), levels[levelMax], in.getSize(), 0);

    for (int l = levelMax - 1; l >= levelMin; --l) {
        runDownsampleMax(levels[l + 1], levels[l], xSize[l + 1], ySize[l + 1], zSize[l + 1], 0);
    }


    // ================== Phase 1 - top to down
    for (int l = levelMin; l <= levelMax; ++l) {
        runOneLevel(levels[l], xSize[l], ySize[l], zSize[l], l, 0);
    }
    // ================== Phase 1 - down to top
    for (int l = levelMax - 1; l >= levelMin; --l) {
        runSecondPhase(levels[l], levels[l+1], xSize[l], ySize[l], zSize[l], xSize[l+1], ySize[l+1], zSize[l+1], l == levelMin, 0);
    }
};
