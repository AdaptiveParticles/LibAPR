#include "ComputeInverseCubicBsplineCuda.h"

#include <hip/hip_runtime.h>
#include <>

#include "invBspline.cuh"
#include "misc/CudaTools.hpp"


namespace {
    void emptyCallForTemplateInstantiation() {
        MeshData<float> f = MeshData<float>(0, 0, 0);
        MeshData<uint16_t> u16 = MeshData<uint16_t>(0, 0, 0);
        MeshData<uint8_t> u8 = MeshData<uint8_t>(0, 0, 0);
        cudaInverseBspline(f);
        cudaInverseBspline(u16);
        cudaInverseBspline(u8);
    }
} 

template <typename ImgType>
void cudaInverseBspline(MeshData<ImgType> &input, TypeOfInvBsplineFlags flags) {
    APRTimer timer(true), timerFullPipelilne(true);
    size_t inputSize = input.mesh.size() * sizeof(ImgType);

    timer.start_timer("cuda: memory alloc + data transfer to device");
    ImgType *cudaInput;
    hipMalloc(&cudaInput, inputSize);
    hipMemcpy(cudaInput, input.mesh.get(), inputSize, hipMemcpyHostToDevice);
    timer.stop_timer();

    constexpr int numOfWorkers = 32;
    timerFullPipelilne.start_timer("cuda: calculations on device FULL <<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<<< ");
    if (flags & INV_BSPLINE_Y_DIR) {
        timer.start_timer("cuda: calculations on device Y ============================================================================ ");
        dim3 threadsPerBlock(1, numOfWorkers, 1);
        dim3 numBlocks((input.x_num + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       1,
                       (input.z_num + threadsPerBlock.z - 1) / threadsPerBlock.z);
        printCudaDims(threadsPerBlock, numBlocks);
        invBsplineYdir<ImgType> <<< numBlocks, threadsPerBlock>>> (cudaInput, input.x_num, input.y_num, input.z_num);
        waitForCuda();
        timer.stop_timer();
    }
    if (flags & INV_BSPLINE_X_DIR) {
        timer.start_timer("cuda: calculations on device X ============================================================================ ");
        dim3 threadsPerBlock(1, numOfWorkers, 1);
        dim3 numBlocks(1,
                       (input.y_num + threadsPerBlock.y - 1) / threadsPerBlock.y,
                       (input.z_num + threadsPerBlock.z - 1) / threadsPerBlock.z);
        printCudaDims(threadsPerBlock, numBlocks);
        invBsplineXdir<ImgType> <<< numBlocks, threadsPerBlock>>> (cudaInput, input.x_num, input.y_num, input.z_num);
        waitForCuda();
        timer.stop_timer();
    }
    if (flags & INV_BSPLINE_Z_DIR) {
        timer.start_timer("cuda: calculations on device Z ============================================================================ ");
        dim3 threadsPerBlock(1, numOfWorkers, 1);
        dim3 numBlocks((input.x_num + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (input.y_num + threadsPerBlock.y - 1) / threadsPerBlock.y,
                       1);
        printCudaDims(threadsPerBlock, numBlocks);
        invBsplineZdir<ImgType> <<< numBlocks, threadsPerBlock>>> (cudaInput, input.x_num, input.y_num, input.z_num);
        waitForCuda();
        timer.stop_timer();
    }
    timerFullPipelilne.stop_timer();

    timer.start_timer("cuda: transfer data from device and freeing memory");
    getDataFromKernel(input, inputSize, cudaInput);
    timer.stop_timer();
}
