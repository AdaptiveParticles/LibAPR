#include "hip/hip_runtime.h"
#include "ComputeGradientCuda.hpp"
#include "APRParameters.hpp"
#include <iostream>
#include <memory>

#include <hip/hip_runtime.h>
#include <>

#include "data_structures/Mesh/PixelData.hpp"
#include "dsGradient.cuh"

#include "invBspline.cuh"
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include "bsplineXdir.cuh"
#include "bsplineYdir.cuh"
#include "bsplineZdir.cuh"
#include "data_structures/Mesh/downsample.cuh"
#include "algorithm/ComputePullingScheme.cuh"
#include "algorithm/LocalIntensityScaleCuda.h"
#include "algorithm/LocalIntensityScale.cuh"
#include "misc/CudaTools.cuh"
#include "misc/CudaMemory.cuh"
#include <chrono>
#include <cstdint>

namespace {
    typedef struct {
        PinnedMemoryUniquePtr<float> bc1;
        PinnedMemoryUniquePtr<float> bc2;
        PinnedMemoryUniquePtr<float> bc3;
        PinnedMemoryUniquePtr<float> bc4;
        size_t k0;
        float b1;
        float b2;
        float norm_factor;
    } BsplineParams;

    float impulse_resp(float k, float rho, float omg) {
        //  Impulse Response Function
        return (pow(rho, (std::abs(k))) * sin((std::abs(k) + 1) * omg)) / sin(omg);
    }

    float impulse_resp_back(float k, float rho, float omg, float gamma, float c0) {
        //  Impulse Response Function (nominator eq. 4.8, denominator from eq. 4.7)
        return c0 * pow(rho, std::abs(k)) * (cos(omg * std::abs(k)) + gamma * sin(omg * std::abs(k))) *
               (1.0 / (pow((1 - 2.0 * rho * cos(omg) + pow(rho, 2)), 2)));
    }

    template<typename T>
    BsplineParams prepareBsplineStuff(const PixelData<T> &image, float lambda, float tol, int maxFilterLen = -1) {
        // Recursive Filter Implimentation for Smoothing BSplines
        // B-Spline Signal Processing: Part II - Efficient Design and Applications, Unser 1993

        float xi = 1 - 96 * lambda + 24 * lambda * sqrt(3 + 144 * lambda); // eq 4.6
        float rho = (24 * lambda - 1 - sqrt(xi)) / (24 * lambda) *
                    sqrt((1 / xi) * (48 * lambda + 24 * lambda * sqrt(3 + 144 * lambda))); // eq 4.5
        float omg = atan(sqrt((1 / xi) * (144 * lambda - 1))); // eq 4.6

        float c0 = (1 + pow(rho, 2)) / (1 - pow(rho, 2)) * (1 - 2 * rho * cos(omg) + pow(rho, 2)) /
                   (1 + 2 * rho * cos(omg) + pow(rho, 2)); // eq 4.8
        float gamma = (1 - pow(rho, 2)) / (1 + pow(rho, 2)) * (1 / tan(omg)); // eq 4.8

        const float b1 = 2 * rho * cos(omg);
        const float b2 = -pow(rho, 2.0);

        const size_t idealK0Len = ceil(std::abs(log(tol) / log(rho)));
        const size_t minDimension = std::min(image.z_num, std::min(image.x_num, image.y_num));
        const size_t k0 = maxFilterLen > 0 ? maxFilterLen : std::min(idealK0Len, minDimension);

        const float norm_factor = pow((1 - 2.0 * rho * cos(omg) + pow(rho, 2)), 2);
        std::cout << "GPU: xi=" << xi << " rho=" << rho << " omg=" << omg << " gamma=" << gamma << " b1=" << b1
                  << " b2=" << b2 << " k0=" << k0 << " norm_factor=" << norm_factor << std::endl;

        // ------- Calculating boundary conditions

        // forward boundaries
        std::vector<float> impulse_resp_vec_f(k0 + 1);
        for (size_t k = 0; k < impulse_resp_vec_f.size(); ++k) impulse_resp_vec_f[k] = impulse_resp(k, rho, omg);

        size_t boundaryLen = sizeof(float) * k0;
        PinnedMemoryUniquePtr<float> bc1{(float*)getPinnedMemory(boundaryLen)};
        PinnedMemoryUniquePtr<float> bc2{(float*)getPinnedMemory(boundaryLen)};
        PinnedMemoryUniquePtr<float> bc3{(float*)getPinnedMemory(boundaryLen)};
        PinnedMemoryUniquePtr<float> bc4{(float*)getPinnedMemory(boundaryLen)};

        //y(0) init
        for (size_t k = 0; k < k0; ++k) bc1[k] = impulse_resp_vec_f[k];
        //y(1) init
        bc2[1] = impulse_resp_vec_f[0];
        for (size_t k = 0; k < k0; ++k) bc2[k] += impulse_resp_vec_f[k + 1];

        // backward boundaries
        std::vector<float> impulse_resp_vec_b(k0 + 1);
        for (size_t k = 0; k < impulse_resp_vec_b.size(); ++k)
            impulse_resp_vec_b[k] = impulse_resp_back(k, rho, omg, gamma, c0);

        //y(N-1) init
        bc3[0] = impulse_resp_vec_b[1];
        for (size_t k = 0; k < (k0 - 1); ++k) bc3[k + 1] += impulse_resp_vec_b[k] + impulse_resp_vec_b[k + 2];
        //y(N) init
        bc4[0] = impulse_resp_vec_b[0];
        for (size_t k = 1; k < k0; ++k) bc4[k] += 2 * impulse_resp_vec_b[k];

        return BsplineParams{
                std::move(bc1),
                std::move(bc2),
                std::move(bc3),
                std::move(bc4),
                k0,
                b1,
                b2,
                norm_factor
        };
    }
}

/**
 * Thresholds output basing on input values. When input is <= thresholdLevel then output is set to 0 and is not changed otherwise.
 * @param input
 * @param output
 * @param length - len of input/output arrays
 * @param thresholdLevel
 */
template <typename T, typename S>
__global__ void threshold(const T *input, S *output, size_t length, float thresholdLevel) {
    size_t idx = (size_t)blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < length) {
        if (input[idx] <= thresholdLevel) { output[idx] = 0; }
    }
}

template <typename ImgType, typename T>
void runThreshold(ImgType *cudaImage, T *cudaGrad, size_t x_num, size_t y_num, size_t z_num, float Ip_th, hipStream_t aStream) {
    dim3 threadsPerBlock(64);
    dim3 numBlocks((x_num * y_num * z_num + threadsPerBlock.x - 1)/threadsPerBlock.x);
    threshold<<<numBlocks,threadsPerBlock, 0, aStream>>>(cudaImage, cudaGrad, x_num * y_num * z_num, Ip_th);
};

/**
 * Thresholds input array to have minimum thresholdLevel.
 * @param input
 * @param length - len of input/output arrays
 * @param thresholdLevel
 */
template <typename T>
__global__ void thresholdImg(T *input, size_t length, float thresholdLevel) {
    size_t idx = (size_t)blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < length) {
        if (input[idx] < thresholdLevel) { input[idx] = thresholdLevel; }
    }
}

template <typename T>
void runThresholdImg(T *cudaImage, size_t x_num, size_t y_num, size_t z_num, float Ip_th_offset, hipStream_t aStream) {
    dim3 threadsPerBlock(64);
    dim3 numBlocks((x_num * y_num * z_num + threadsPerBlock.x - 1) / threadsPerBlock.x);
    thresholdImg<<< numBlocks, threadsPerBlock, 0, aStream >>> (cudaImage, x_num * y_num * z_num, Ip_th_offset);
};

template <typename ImgType>
void getGradientCuda(const PixelData<ImgType> &image, PixelData<float> &local_scale_temp,
                     ImgType *cudaImage, ImgType *cudaGrad, float *cudalocal_scale_temp,
                     BsplineParams &p, float *bc1, float *bc2, float *bc3, float *bc4, float *boundary,
                     float bspline_offset, const APRParameters &par, hipStream_t aStream) {

    runThresholdImg(cudaImage, image.x_num, image.y_num, image.z_num, par.Ip_th + bspline_offset, aStream);

    runBsplineYdir(cudaImage, image.x_num, image.y_num, image.z_num, bc1, bc2, bc3, bc4, p.k0, p.b1, p.b2, p.norm_factor, boundary, aStream);
    runBsplineXdir(cudaImage, image.x_num, image.y_num, image.z_num, bc1, bc2, bc3, bc4, p.k0, p.b1, p.b2, p.norm_factor, aStream);
    runBsplineZdir(cudaImage, image.x_num, image.y_num, image.z_num, bc1, bc2, bc3, bc4, p.k0, p.b1, p.b2, p.norm_factor, aStream);

    runKernelGradient(cudaImage, cudaGrad, image.x_num, image.y_num, image.z_num, local_scale_temp.x_num, local_scale_temp.y_num, par.dx, par.dy, par.dz, aStream);

    runDownsampleMean(cudaImage, cudalocal_scale_temp, image.x_num, image.y_num, image.z_num, aStream);

    runInvBsplineYdir(cudalocal_scale_temp, local_scale_temp.x_num, local_scale_temp.y_num, local_scale_temp.z_num, aStream);
    runInvBsplineXdir(cudalocal_scale_temp, local_scale_temp.x_num, local_scale_temp.y_num, local_scale_temp.z_num, aStream);
    runInvBsplineZdir(cudalocal_scale_temp, local_scale_temp.x_num, local_scale_temp.y_num, local_scale_temp.z_num, aStream);

    runThreshold(cudalocal_scale_temp, cudaGrad, local_scale_temp.x_num, local_scale_temp.y_num, local_scale_temp.z_num, par.Ip_th, aStream);
}

class CurrentTime {
    std::chrono::high_resolution_clock m_clock;

public:
    uint64_t milliseconds() {
        return std::chrono::duration_cast<std::chrono::milliseconds>
                (m_clock.now().time_since_epoch()).count();
    }
    uint64_t microseconds() {
        return std::chrono::duration_cast<std::chrono::microseconds>
                (m_clock.now().time_since_epoch()).count();
    }
    uint64_t nanoseconds() {
        return std::chrono::duration_cast<std::chrono::nanoseconds>
                (m_clock.now().time_since_epoch()).count();
    }
};

template <typename U>
template <typename ImgType>
class GpuProcessingTask<U>::GpuProcessingTaskImpl {

    // input data
    const PixelData<ImgType> &iCpuImage;
    PixelData<float> &iCpuLevels;
    const APRParameters &iParameters;
    float iBsplineOffset;
    int iMaxLevel;

    // cuda stuff - memory and stream to be used
    const hipStream_t iStream;
    ScopedCudaMemHandler<const PixelData<ImgType>, JUST_ALLOC> image;
    ScopedCudaMemHandler<PixelData<ImgType>, JUST_ALLOC> gradient;
    ScopedCudaMemHandler<PixelData<float>, JUST_ALLOC> local_scale_temp;
    ScopedCudaMemHandler<PixelData<float>, JUST_ALLOC> local_scale_temp2;

    // bspline stuff
    const float tolerance = 0.0001;
    BsplineParams params;
    ScopedCudaMemHandler<float*, H2D> bc1;
    ScopedCudaMemHandler<float*, H2D> bc2;
    ScopedCudaMemHandler<float*, H2D> bc3;
    ScopedCudaMemHandler<float*, H2D> bc4;
    const size_t boundaryLen;
    ScopedCudaMemHandler<float*, JUST_ALLOC> boundary;

    /**
     * @return newly created stream
     */
    hipStream_t getStream() {
        hipStream_t stream;
        hipStreamCreate(&stream);
        return stream;
    }

public:

    GpuProcessingTaskImpl(const PixelData<ImgType> &image, PixelData<float> &levels, const APRParameters &parameters, float bspline_offset, int maxLevel) :
        iCpuImage(image),
        iCpuLevels(levels),
        iStream(getStream()),
        image (image, iStream),
        gradient (levels, iStream),
        local_scale_temp (levels, iStream),
        local_scale_temp2 (levels, iStream),
        iParameters(parameters),
        iBsplineOffset(bspline_offset),
        iMaxLevel(maxLevel),
        params(prepareBsplineStuff(image, parameters.lambda, tolerance)),
        bc1(params.bc1.get(), params.k0, iStream),
        bc2(params.bc2.get(), params.k0, iStream),
        bc3(params.bc3.get(), params.k0, iStream),
        bc4(params.bc4.get(), params.k0, iStream),
        boundaryLen{(2 /*two first elements*/ + 2 /* two last elements */) * image.x_num * image.z_num},
        boundary{nullptr, boundaryLen, iStream}
    {
//        std::cout << "\n=============== GpuProcessingTaskImpl ===================\n\n";
        std::cout << iCpuImage << std::endl;
        std::cout << iCpuLevels << std::endl;
        std::cout << "\n\n\n";

    }

    void sendDataToGpu() {
        CurrentTime ct;
        uint64_t start = ct.microseconds();
        image.copyH2D();
        std::cout << "SEND time: " << ct.microseconds() - start << std::endl;
    }

    void getDataFromGpu() {
        CurrentTime ct;
        uint64_t start = ct.microseconds();
        local_scale_temp.copyD2H();
        hipStreamSynchronize(iStream);
        std::cout << "RCV time: " << ct.microseconds() - start << std::endl;
    }

    void processOnGpu() {
        CurrentTime ct;
        uint64_t start = ct.microseconds();
        getGradientCuda(iCpuImage, iCpuLevels, image.get(), gradient.get(), local_scale_temp.get(),
                        params, bc1.get(), bc2.get(), bc3.get(), bc4.get(), boundary.get(),
                        iBsplineOffset, iParameters, iStream);
        std::cout << "1: " << ct.microseconds() - start << std::endl;
        runLocalIntensityScalePipeline(iCpuLevels, iParameters, local_scale_temp.get(), local_scale_temp2.get(), iStream);
        std::cout << "2: " << ct.microseconds() - start << std::endl;
        float min_dim = std::min(iParameters.dy, std::min(iParameters.dx, iParameters.dz));
        float level_factor = pow(2, iMaxLevel) * min_dim;
        const float mult_const = level_factor/iParameters.rel_error;
        runComputeLevels(gradient.get(), local_scale_temp.get(), iCpuLevels.mesh.size(), mult_const, iStream);
        std::cout << "3: " << ct.microseconds() - start << std::endl;
    }

    ~GpuProcessingTaskImpl() {
        hipStreamDestroy(iStream);
//        std::cout << "\n============== ~GpuProcessingTaskImpl ===================\n\n";
    }
};

template <typename ImgType>
GpuProcessingTask<ImgType>::GpuProcessingTask(PixelData<ImgType> &image, PixelData<float> &levels, const APRParameters &parameters, float bspline_offset, int maxLevel)
: impl{new GpuProcessingTaskImpl<ImgType>(image, levels, parameters, bspline_offset, maxLevel)} {std::cout << "GpuProcessingTask\n";}

template <typename ImgType>
GpuProcessingTask<ImgType>::~GpuProcessingTask() {std::cout << "~GpuProcessingTask\n";}

template <typename ImgType>
GpuProcessingTask<ImgType>::GpuProcessingTask(GpuProcessingTask&&) = default;

template <typename ImgType>
void GpuProcessingTask<ImgType>::sendDataToGpu() {impl->sendDataToGpu();}

template <typename ImgType>
void GpuProcessingTask<ImgType>::getDataFromGpu() {impl->getDataFromGpu();}

template <typename ImgType>
void GpuProcessingTask<ImgType>::processOnGpu() {impl->processOnGpu();}

template <typename ImgType>
void GpuProcessingTask<ImgType>::doAll() {
    sendDataToGpu();
    processOnGpu();
    getDataFromGpu();
}

// explicit instantiation of handled types
template class GpuProcessingTask<uint16_t>;
template class GpuProcessingTask<float>;

// ================================== TEST helpers ==============
// TODO: should be moved somewhere

// explicit instantiation of handled types
template void cudaFilterBsplineFull(PixelData<float> &, float, float, TypeOfRecBsplineFlags, int);
template <typename ImgType>
void cudaFilterBsplineFull(PixelData<ImgType> &input, float lambda, float tolerance, TypeOfRecBsplineFlags flags, int maxFilterLen) {
    hipStream_t  aStream = 0;

    BsplineParams p = prepareBsplineStuff(input, lambda, tolerance, maxFilterLen);
    ScopedCudaMemHandler<float*, H2D> bc1(p.bc1.get(), p.k0);
    ScopedCudaMemHandler<float*, H2D> bc2(p.bc2.get(), p.k0);
    ScopedCudaMemHandler<float*, H2D> bc3(p.bc3.get(), p.k0);
    ScopedCudaMemHandler<float*, H2D> bc4(p.bc4.get(), p.k0);
    ScopedCudaMemHandler<PixelData<ImgType>, D2H | H2D> cudaInput(input);

    if (flags & BSPLINE_Y_DIR) {
        int boundaryLen = (2 /*two first elements*/ + 2 /* two last elements */) * input.x_num * input.z_num;
        ScopedCudaMemHandler<float*, JUST_ALLOC> boundary(nullptr, boundaryLen); // allocate memory on device
        runBsplineYdir(cudaInput.get(), input.x_num, input.y_num, input.z_num, bc1.get(), bc2.get(), bc3.get(), bc4.get(), p.k0, p.b1, p.b2, p.norm_factor, boundary.get(), aStream);
    }
    if (flags & BSPLINE_X_DIR) {
        runBsplineXdir(cudaInput.get(), input.x_num, input.y_num, input.z_num, bc1.get(), bc2.get(), bc3.get(), bc4.get(), p.k0, p.b1, p.b2, p.norm_factor, aStream);
    }
    if (flags & BSPLINE_Z_DIR) {
        runBsplineZdir(cudaInput.get(), input.x_num, input.y_num, input.z_num, bc1.get(), bc2.get(), bc3.get(), bc4.get(), p.k0, p.b1, p.b2, p.norm_factor, aStream);
    }
}

// explicit instantiation of handled types
template void cudaInverseBspline(PixelData<float> &, TypeOfInvBsplineFlags);
template <typename ImgType>
void cudaInverseBspline(PixelData<ImgType> &input, TypeOfInvBsplineFlags flags) {
    ScopedCudaMemHandler<PixelData<ImgType>, H2D | D2H> cudaInput(input);

    if (flags & INV_BSPLINE_Y_DIR) {
        runInvBsplineYdir(cudaInput.get(), input.x_num, input.y_num, input.z_num, 0);
    }
    if (flags & INV_BSPLINE_X_DIR) {
        runInvBsplineXdir(cudaInput.get(), input.x_num, input.y_num, input.z_num, 0);
    }
    if (flags & INV_BSPLINE_Z_DIR) {
        runInvBsplineZdir(cudaInput.get(), input.x_num, input.y_num, input.z_num, 0);
    }
}

// explicit instantiation of handled types
template void computeLevelsCuda(const PixelData<float> &, PixelData<float> &, int, float, float, float, float);
template <typename ImageType>
void computeLevelsCuda(const PixelData<ImageType> &grad_temp, PixelData<float> &local_scale_temp, int maxLevel, float relError,  float dx, float dy, float dz) {
    ScopedCudaMemHandler<const PixelData<ImageType>, H2D> cudaGrad(grad_temp);
    ScopedCudaMemHandler<PixelData<float>, D2H | H2D> cudaLis(local_scale_temp);

    float min_dim = std::min(dy, std::min(dx, dz));
    float level_factor = pow(2, maxLevel) * min_dim;
    const float mult_const = level_factor/relError;
    hipStream_t aStream = 0;
    runComputeLevels(cudaGrad.get(), cudaLis.get(), grad_temp.mesh.size(), mult_const, aStream);
}

// explicit instantiation of handled types
template void getGradient(PixelData<float> &, PixelData<float> &, PixelData<float> &, PixelData<float> &, float, const APRParameters &);
template <typename ImgType>
void getGradient(PixelData<ImgType> &image, PixelData<ImgType> &grad_temp, PixelData<float> &local_scale_temp, PixelData<float> &local_scale_temp2, float bspline_offset, const APRParameters &par) {
    ScopedCudaMemHandler<PixelData<ImgType>, D2H | H2D> cudaImage(image);
    ScopedCudaMemHandler<PixelData<ImgType>, D2H | H2D> cudaGrad(grad_temp);
    ScopedCudaMemHandler<PixelData<float>, D2H> cudalocal_scale_temp(local_scale_temp);
    ScopedCudaMemHandler<PixelData<float>, D2H> cudalocal_scale_temp2(local_scale_temp2);

    float tolerance = 0.0001;
    BsplineParams p = prepareBsplineStuff(image, par.lambda, tolerance);

    ScopedCudaMemHandler<float*, H2D> bc1 (p.bc1.get(), p.k0);
    ScopedCudaMemHandler<float*, H2D> bc2 (p.bc2.get(), p.k0);
    ScopedCudaMemHandler<float*, H2D> bc3 (p.bc3.get(), p.k0);
    ScopedCudaMemHandler<float*, H2D> bc4 (p.bc4.get(), p.k0);
    int boundaryLen = (2 /*two first elements*/ + 2 /* two last elements */) * image.x_num * image.z_num;
    ScopedCudaMemHandler<float*, JUST_ALLOC> boundary(nullptr, boundaryLen);

    getGradientCuda(image, local_scale_temp, cudaImage.get(), cudaGrad.get(), cudalocal_scale_temp.get(),
                    p, bc1.get(), bc2.get(), bc3.get(), bc4.get(), boundary.get(),
                    bspline_offset, par, 0);
}

// explicit instantiation of handled types
template void thresholdImg(PixelData<float> &, const float);
template <typename T>
void thresholdImg(PixelData<T> &image, const float threshold) {
    ScopedCudaMemHandler<PixelData<T>, H2D | D2H> cudaImage(image);

    runThresholdImg(cudaImage.get(), image.x_num, image.y_num, image.z_num, threshold, 0);
}

// explicit instantiation of handled types
template void thresholdGradient(PixelData<float> &, const PixelData<float> &, const float);
template <typename T>
void thresholdGradient(PixelData<float> &output, const PixelData<T> &input, const float Ip_th) {
    ScopedCudaMemHandler<const PixelData<T>, H2D> cudaInput(input);
    ScopedCudaMemHandler<PixelData<float>, H2D | D2H> cudaOutput(output);

    runThreshold(cudaInput.get(), cudaOutput.get(), input.x_num, input.y_num, input.z_num, Ip_th, 0);
}

void cudaDownsampledGradient(PixelData<float> &input, PixelData<float> &grad, const float hx, const float hy, const float hz) {
    ScopedCudaMemHandler<PixelData<float>, H2D | D2H> cudaInput(input);
    ScopedCudaMemHandler<PixelData<float>, D2H> cudaGrad(grad);

    runKernelGradient(cudaInput.get(), cudaGrad.get(), input.x_num, input.y_num, input.z_num, grad.x_num, grad.y_num, hx, hy, hz, 0);
}
