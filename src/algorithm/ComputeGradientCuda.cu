#include "hip/hip_runtime.h"
#include "ComputeGradientCuda.hpp"
#include <iostream>
#include <memory>

#include <hip/hip_runtime.h>
#include <>

#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>


__global__ void gradient(float *input, size_t x_num, size_t y_num, size_t z_num, float *grad, size_t x_num_ds, size_t y_num_ds, float hx, float hy, float hz) {
    const int xi = ((blockIdx.x * blockDim.x) + threadIdx.x) * 2;
    const int yi = ((blockIdx.y * blockDim.y) + threadIdx.y) * 2;
    const int zi = ((blockIdx.z * blockDim.z) + threadIdx.z) * 2;
    if (xi >= x_num || yi >= y_num || zi >= z_num) return;

    const size_t xnumynum = x_num * y_num;

    float temp[4][4][4];

    for (int z = 0; z < 4; ++z)
        for (int x = 0; x < 4; ++x)
            for(int y = 0; y < 4; ++y) {
                int xc = xi + x - 1; if (xc < 0) xc = 0; else if (xc >= x_num) xc = x_num - 1;
                int yc = yi + y - 1; if (yc < 0) yc = 0; else if (yc >= y_num) yc = y_num - 1;
                int zc = zi + z - 1; if (zc < 0) zc = 0; else if (zc >= z_num) zc = z_num - 1;
                temp[z][x][y] = *(input + zc * xnumynum + xc * y_num + yc);
            }
    float maxGrad = 0;
    for (int z = 1; z <= 2; ++z)
        for (int x = 1; x <= 2; ++x)
            for(int y = 1; y <= 2; ++y) {
                float xd = (temp[z][x-1][y] - temp[z][x+1][y]) / (2 * hx); xd = xd * xd;
                float yd = (temp[z-1][x][y] - temp[z+1][x][y]) / (2 * hy); yd = yd * yd;
                float zd = (temp[z][x][y-1] - temp[z][x][y+1]) / (2 * hz); zd = zd * zd;
                float gm = __fsqrt_rn(xd + yd + zd);
                if (gm > maxGrad)  maxGrad = gm;
            }

    const size_t idx = zi/2 * x_num_ds * y_num_ds + xi/2 * y_num_ds + yi/2;
    grad[idx] = maxGrad;
}

void cudaDownsampledGradient(const MeshData<float> &input, MeshData<float> &grad, const float hx, const float hy,const float hz) {
    APRTimer timer;
    timer.verbose_flag=true;

    timer.start_timer("cuda: memory alloc + data transfer to device");
    size_t inputSize = input.mesh.size() * sizeof(float);
    float *cudaInput;
    hipMalloc(&cudaInput, inputSize);
    hipMemcpy(cudaInput, input.mesh.get(), inputSize, hipMemcpyHostToDevice);

    size_t gradSize = grad.mesh.size() * sizeof(float);
    float *cudaGrad;
    hipMalloc(&cudaGrad, gradSize);
    timer.stop_timer();

    timer.start_timer("cuda: calculations on device");
    dim3 threadsPerBlock(1, 32, 1);
    dim3 numBlocks((input.x_num + threadsPerBlock.x - 1)/threadsPerBlock.x,
                   (input.y_num + threadsPerBlock.y - 1)/threadsPerBlock.y,
                   (input.z_num + threadsPerBlock.z - 1)/threadsPerBlock.z);
    std::cout << "Number of blocks  (x/y/z):  " << numBlocks.x << "/" << numBlocks.y << "/" << numBlocks.z << std::endl;
    std::cout << "Number of threads (x/y/z): " << threadsPerBlock.x << "/" << threadsPerBlock.y << "/" << threadsPerBlock.z << std::endl;

    gradient<<<numBlocks,threadsPerBlock>>>(cudaInput, input.x_num, input.y_num, input.z_num, cudaGrad, grad.x_num, grad.y_num, hx, hy, hz);
    hipDeviceSynchronize();
    timer.stop_timer();

    timer.start_timer("cuda: transfer data from device and freeing memory");
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)printf("Error: %s\n", hipGetErrorString(err));
    hipMemcpy((void*)input.mesh.get(), cudaInput, inputSize, hipMemcpyDeviceToHost);
    hipFree(cudaInput);
    hipMemcpy((void*)grad.mesh.get(), cudaGrad, gradSize, hipMemcpyDeviceToHost);
    hipFree(cudaGrad);
    timer.stop_timer();
}

/////////////////////////////////////////////////////////////////////


float impulse_resp(float k,float rho,float omg){
    //  Impulse Response Function
    return (pow(rho,(std::abs(k)))*sin((std::abs(k) + 1)*omg)) / sin(omg);
}

float impulse_resp_back(float k,float rho,float omg,float gamma,float c0){
    //  Impulse Response Function (nominator eq. 4.8, denominator from eq. 4.7)
    return c0*pow(rho,std::abs(k))*(cos(omg*std::abs(k)) + gamma*sin(omg*std::abs(k)))*(1.0/(pow((1 - 2.0*rho*cos(omg) + pow(rho,2)),2)));
}


typedef struct {
    std::vector<float> bc1_vec;
    std::vector<float> bc2_vec;
    std::vector<float> bc3_vec;
    std::vector<float> bc4_vec;
    size_t k0;
    float b1;
    float b2;
    float norm_factor;
} BsplineParams;

template <typename T>
BsplineParams prepareBsplineStuff(MeshData<T> & image, float lambda, float tol) {
    float xi = 1 - 96*lambda + 24*lambda*sqrt(3 + 144*lambda); // eq 4.6
    float rho = (24*lambda - 1 - sqrt(xi))/(24*lambda)*sqrt((1/xi)*(48*lambda + 24*lambda*sqrt(3 + 144*lambda))); // eq 4.5
    float omg = atan(sqrt((1/xi)*(144*lambda - 1))); // eq 4.6

    float c0 = (1+ pow(rho,2))/(1-pow(rho,2)) * (1 - 2*rho*cos(omg) + pow(rho,2))/(1 + 2*rho*cos(omg) + pow(rho,2)); // eq 4.8
    float gamma = (1-pow(rho,2))/(1+pow(rho,2)) * (1/tan(omg)); // eq 4.8

    const float b1 = 2*rho*cos(omg);
    const float b2 = -pow(rho,2.0);

    const size_t z_num = image.z_num;
    const size_t xxx = ceil(std::abs(log(tol)/log(rho)));
    const size_t k0 = std::min(xxx, z_num);

    const float norm_factor = pow((1 - 2.0*rho*cos(omg) + pow(rho,2)),2);
    std::cout << "GPU: " << xi << " " << rho << " " << omg << " " << gamma << " " << b1 << " " << b2 << " " << k0 << " " << norm_factor << std::endl;
    //////////////////////////////////////////////////////////////
    //
    //  Setting up boundary conditions
    //
    //////////////////////////////////////////////////////////////

    // for boundaries
    std::cout << "k0=" << k0 << std::endl;
    std::vector<float> impulse_resp_vec_f(k0+3);  //forward
    for (size_t k = 0; k < (k0+3); ++k) {
        impulse_resp_vec_f[k] = impulse_resp(k,rho,omg);
    }

    std::vector<float> impulse_resp_vec_b(k0+3);  //backward
    for (size_t k = 0; k < (k0+3); ++k) {
        impulse_resp_vec_b[k] = impulse_resp_back(k,rho,omg,gamma,c0);
    }

    std::vector<float> bc1_vec(k0, 0);  //forward
    //y(1) init
    bc1_vec[1] = impulse_resp_vec_f[0];
    for (size_t k = 0; k < k0; ++k) {
        bc1_vec[k] += impulse_resp_vec_f[k+1];
    }

    std::vector<float> bc2_vec(k0, 0);  //backward
    //y(0) init
    for (size_t k = 0; k < k0; ++k) {
        bc2_vec[k] = impulse_resp_vec_f[k];
    }

    std::vector<float> bc3_vec(k0, 0);  //forward
    //y(N-1) init
    bc3_vec[0] = impulse_resp_vec_b[1];
    for (size_t k = 0; k < (k0-1); ++k) {
        bc3_vec[k+1] += impulse_resp_vec_b[k] + impulse_resp_vec_b[k+2];
    }

    std::vector<float> bc4_vec(k0, 0);  //backward
    //y(N) init
    bc4_vec[0] = impulse_resp_vec_b[0];
    for (size_t k = 1; k < k0; ++k) {
        bc4_vec[k] += 2*impulse_resp_vec_b[k];
    }

    return BsplineParams {
            bc1_vec,
            bc2_vec,
            bc3_vec,
            bc4_vec,
            k0,
            b1,
            b2,
            norm_factor
    };
}

 //========== First naive version following CPU code ===============
template <typename T>
__global__ void bsplineY(T *image, size_t x_num, size_t y_num, size_t z_num, float *bc1_vec, float *bc2_vec, float *bc3_vec, float *bc4_vec, size_t k0, float b1, float b2, float norm_factor) {
     int xi = ((blockIdx.x * blockDim.x) + threadIdx.x);
     int zi = ((blockIdx.z * blockDim.z) + threadIdx.z);
     if (xi >= x_num || zi >= z_num) return;

     //forwards direction
     const size_t zPlaneOffset = zi * x_num * y_num;
     const size_t yColOffset = xi * y_num;
     size_t yCol = zPlaneOffset + yColOffset;

     float temp1 = 0;
     float temp2 = 0;
     float temp3 = 0;
     float temp4 = 0;

     for (size_t k = 0; k < k0; ++k) {
         temp1 += bc1_vec[k]*image[yCol + k];
         temp2 += bc2_vec[k]*image[yCol + k];
         temp3 += bc3_vec[k]*image[yCol + y_num - 1 - k];
         temp4 += bc4_vec[k]*image[yCol + y_num - 1 - k];
     }

     //initialize the sequence
     image[yCol + 0] = temp2;
     image[yCol + 1] = temp1;

     // middle values
     for (auto it = (image + yCol + 2); it !=  (image+yCol + y_num); ++it) {
         float  temp = temp1*b1 + temp2*b2 + *it;
         *it = temp;
         temp2 = temp1;
         temp1 = temp;
     }

     // finish sequence
     image[yCol + y_num - 2] = temp3;
     image[yCol + y_num - 1] = temp4;

     // -------------- part 2
     temp2 = image[yCol + y_num - 1];
     temp1 = image[yCol + y_num - 2];
     image[yCol + y_num - 1]*=norm_factor;
     image[yCol + y_num - 2]*=norm_factor;

     for (auto it = (image + yCol + y_num-3); it !=  (image + yCol - 1); --it) {
         float temp = temp1*b1 + temp2*b2 + *it;
         *it = temp*norm_factor;
         temp2 = temp1;
         temp1 = temp;
     }
}

extern __shared__ float sharedMem[];
template<typename T>
__global__ void bsplineYdirBoundary(T *image, size_t x_num, size_t y_num, size_t z_num,
                                    const float *bc1_vec, const float *bc2_vec, const float *bc3_vec, const float *bc4_vec,
                                    size_t k0, float *boundary) {
    const int xzIndexOfWorker = (blockIdx.x * blockDim.x) + threadIdx.x;
    const int xzIndexOfBlock = (blockIdx.x * blockDim.x);

    const int numOfWorkers = blockDim.x;
    const int currentWorkerId = threadIdx.x;
    const size_t workersOffset = xzIndexOfBlock * y_num; // per each (x,z) coordinate we have y-row

    const int64_t maxXZoffset = x_num * z_num;

    float *bc1_vec2 = &sharedMem[0];
    float *bc2_vec2 = &bc1_vec2[k0];
    T *cache = (T*)&bc2_vec2[k0];

    // Read from global mem to cache
    for (int i = currentWorkerId; i < k0 * numOfWorkers; i += numOfWorkers) {
        if (i < k0) {
            bc1_vec2[i] = bc1_vec[i];
            bc2_vec2[i] = bc2_vec[i];
        }
        int offs = i % k0;
        int work = i / k0;
        if (work + xzIndexOfBlock < maxXZoffset) {
            cache[work * k0 + offs] = image[workersOffset + y_num * work + offs];
        }
    }
    __syncthreads();

    //forwards direction
    if (xzIndexOfWorker < x_num * z_num) {
        float temp1 = 0;
        float temp2 = 0;
        for (size_t k = 0; k < k0; ++k) {
            temp1 += bc1_vec2[k] * cache[currentWorkerId * k0 + k];
            temp2 += bc2_vec2[k] * cache[currentWorkerId * k0 + k];
        }
        boundary[xzIndexOfWorker*4 + 0] = temp2;
        boundary[xzIndexOfWorker*4 + 1] = temp1;
    }

    // ----------------- second end
    __syncthreads();

    for (int i = currentWorkerId; i < k0 * numOfWorkers; i += numOfWorkers) {
        if (i < k0) {
            bc1_vec2[i] = bc3_vec[i];
            bc2_vec2[i] = bc4_vec[i];
        }
        int offs = i % k0;
        int work = i / k0;
        if (work + xzIndexOfBlock < maxXZoffset) {
            cache[work * k0 + offs] = image[workersOffset + y_num * work + y_num - 1 - offs];
        }
    }
    __syncthreads();

    //forwards direction
    if (xzIndexOfWorker < x_num * z_num) {
        float temp3 = 0;
        float temp4 = 0;
        for (size_t k = 0; k < k0; ++k) {
            temp3 += bc1_vec2[k] * cache[currentWorkerId * k0 + k];
            temp4 += bc2_vec2[k] * cache[currentWorkerId * k0 + k];
        }
        boundary[xzIndexOfWorker*4 + 2] = temp3;
        boundary[xzIndexOfWorker*4 + 3] = temp4;
    }
}

constexpr int blockWidth = 32;
constexpr int numOfThreads = 32;
extern __shared__ char sharedMemProcess[];
template<typename T>
__global__ void bsplineYdirProcess(T *image, const size_t x_num, const size_t y_num, const size_t z_num, size_t k0,
                                   const float b1, const float b2, const float norm_factor, float *boundary) {
    const int numOfWorkers = blockDim.x;
    const int currentWorkerId = threadIdx.x;
    const int xzOffset = blockIdx.x * blockDim.x;
    const int64_t maxXZoffset = x_num * z_num;
    const int64_t workersOffset = xzOffset * y_num;

    T (*cache)[blockWidth + 0] = (T (*)[blockWidth + 0]) &sharedMemProcess[0];

    float temp1, temp2;

    // ---------------- forward direction -------------------------------------------
    for (int yBlockBegin = 0; yBlockBegin < y_num - 2; yBlockBegin += blockWidth) {

        // Read from global mem to cache
        for (int i = currentWorkerId; i < blockWidth * numOfWorkers; i += numOfWorkers) {
            int offs = i % blockWidth;
            int work = i / blockWidth;
            if (offs + yBlockBegin < (y_num - 2) && work + xzOffset < maxXZoffset) {
                cache[work][(offs + work)%blockWidth] = image[workersOffset + y_num * work + offs + yBlockBegin];
            }
        }
        __syncthreads();

        // Do operations
        if (xzOffset + currentWorkerId < maxXZoffset) {
            if (yBlockBegin == 0) {
                temp2 = boundary[(xzOffset + currentWorkerId) * 4 + 0];
                temp1 = boundary[(xzOffset + currentWorkerId) * 4 + 1];
                cache[currentWorkerId][(0 + currentWorkerId)%blockWidth] = temp2;
                cache[currentWorkerId][(1 + currentWorkerId)%blockWidth] = temp1;
            }
            for (size_t k = yBlockBegin == 0 ? 2 : 0; k < blockWidth && k + yBlockBegin < y_num - 2; ++k) {
                float  temp = temp1*b1 + temp2*b2 + cache[currentWorkerId][(k + currentWorkerId)%blockWidth];
                cache[currentWorkerId][(k + currentWorkerId)%blockWidth] = temp;
                temp2 = temp1;
                temp1 = temp;
            }
        }
        __syncthreads();

        // Write from cache to global mem
        for (int i = currentWorkerId; i < blockWidth * numOfWorkers; i += numOfWorkers) {
            int offs = i % blockWidth;
            int work = i / blockWidth;
            if (offs + yBlockBegin < (y_num - 2) && work + xzOffset < maxXZoffset) {
                image[workersOffset + y_num * work + offs + yBlockBegin] = cache[work][(offs + work)%blockWidth];
            }
        }
        __syncthreads();
    }

    // ---------------- backward direction -------------------------------------------
    for (int yBlockBegin = y_num - 1; yBlockBegin >= 0; yBlockBegin -= blockWidth) {

        // Read from global mem to cache
        for (int i = currentWorkerId; i < blockWidth * numOfWorkers; i += numOfWorkers) {
            int offs = i % blockWidth;
            int work = i / blockWidth;
            if (yBlockBegin - offs >= 0 && work + xzOffset < maxXZoffset) {
                cache[work][(offs + work)%blockWidth] = image[workersOffset + y_num * work - offs + yBlockBegin];
            }
        }
        __syncthreads();

        // Do operations
        if (xzOffset + currentWorkerId < maxXZoffset) {
            if (yBlockBegin == y_num - 1) {
                temp2 = boundary[(xzOffset + currentWorkerId) * 4 + 3];
                temp1 = boundary[(xzOffset + currentWorkerId) * 4 + 2];
                cache[currentWorkerId][(0 + currentWorkerId)%blockWidth] = norm_factor * temp2;
                cache[currentWorkerId][(1 + currentWorkerId)%blockWidth] = norm_factor * temp1;
            }
            for (int64_t k = yBlockBegin == y_num - 1 ? 2 : 0; k < blockWidth && yBlockBegin - k >= 0; ++k) {
                float  temp = temp1*b1 + temp2*b2 + cache[currentWorkerId][(k + currentWorkerId)%blockWidth];
                cache[currentWorkerId][(k + currentWorkerId)%blockWidth] = temp * norm_factor;
                temp2 = temp1;
                temp1 = temp;
            }
        }
        __syncthreads();

        // Write from cache to global mem
        for (int i = currentWorkerId; i < blockWidth * numOfWorkers; i += numOfWorkers) {
            int offs = i % blockWidth;
            int work = i / blockWidth;
            if (yBlockBegin - offs >= 0 && work + xzOffset < maxXZoffset) {
                image[workersOffset + y_num * work - offs + yBlockBegin] = cache[work][(offs + work)%blockWidth];
            }
        }
        __syncthreads();
    }
}

template <typename ImgType>
void cudaFilterBsplineYdirection(MeshData<ImgType> &input, float lambda, float tolerance) {
    APRTimer timer;
    timer.verbose_flag=true;

    BsplineParams p = prepareBsplineStuff(input, lambda, tolerance);

    timer.start_timer("cuda: memory alloc + data transfer to device");
    size_t inputSize = input.mesh.size() * sizeof(ImgType);
    ImgType *cudaInput;
    hipMalloc(&cudaInput, inputSize);
    hipMemcpy(cudaInput, input.mesh.get(), inputSize, hipMemcpyHostToDevice);
    float *boundary;
    int boundaryLen = sizeof(float) * 4 * input.x_num * input.z_num;
    hipMalloc(&boundary, boundaryLen);

    thrust::device_vector<float> d_bc1_vec(p.bc1_vec);
    thrust::device_vector<float> d_bc2_vec(p.bc2_vec);
    thrust::device_vector<float> d_bc3_vec(p.bc3_vec);
    thrust::device_vector<float> d_bc4_vec(p.bc4_vec);
    timer.stop_timer();

    hipDeviceSetCacheConfig(hipFuncCachePreferShared);
    hipDeviceSetSharedMemConfig(hipSharedMemBankSizeFourByte);

    dim3 threadsPerBlock(numOfThreads, 1, 1);
    dim3 numBlocks((input.x_num * input.z_num + threadsPerBlock.x - 1)/threadsPerBlock.x,
                   1,
                   1);
    std::cout << "Number of blocks  (x/y/z):  " << numBlocks.x << "/" << numBlocks.y << "/" << numBlocks.z << std::endl;
    std::cout << "Number of threads (x/y/z): " << threadsPerBlock.x << "/" << threadsPerBlock.y << "/" << threadsPerBlock.z << std::endl;

    float *bc1 = thrust::raw_pointer_cast(d_bc1_vec.data());
    float *bc2 = thrust::raw_pointer_cast(d_bc2_vec.data());
    float *bc3 = thrust::raw_pointer_cast(d_bc3_vec.data());
    float *bc4 = thrust::raw_pointer_cast(d_bc4_vec.data());

    timer.start_timer("cuda: calculations on device ============================================================================ ");
    if (true) {
        bsplineYdirBoundary<ImgType> << < numBlocks, threadsPerBlock,
                (2 /*bc vectors*/) * (p.k0) * sizeof(float) + numOfThreads * (p.k0) * sizeof(ImgType) >> >
                (cudaInput, input.x_num, input.y_num, input.z_num, bc1, bc2, bc3, bc4, p.k0, boundary);
        float *boundaryHost = new float[boundaryLen]; //TODO: free it
        hipMemcpy(boundaryHost, boundary, boundaryLen, hipMemcpyDeviceToHost);
        bsplineYdirProcess<ImgType> << < numBlocks, threadsPerBlock,
                numOfThreads * (0 + blockWidth) * sizeof(ImgType) >> >
                (cudaInput, input.x_num, input.y_num, input.z_num, p.k0, p.b1, p.b2, p.norm_factor, boundary);
        delete[] boundaryHost;
    } else {
        // old naive approach
        dim3 threadsPerBlock(8, 1, 8);
        dim3 numBlocks((input.x_num + threadsPerBlock.x - 1)/threadsPerBlock.x,
                       1,
                       (input.z_num + threadsPerBlock.z - 1)/threadsPerBlock.z);
        bsplineY<ImgType> <<<numBlocks, threadsPerBlock>>>(cudaInput, input.x_num, input.y_num, input.z_num, bc1, bc2, bc3, bc4, p.k0, p.b1, p.b2, p.norm_factor);
    }

    hipDeviceSynchronize();

    timer.stop_timer();

    timer.start_timer("cuda: transfer data from device and freeing memory");
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)printf("Error: %s\n", hipGetErrorString(err));

    hipMemcpy((void*)input.mesh.get(), cudaInput, inputSize, hipMemcpyDeviceToHost);
    hipFree(cudaInput);
    timer.stop_timer();
}


constexpr int numOfWorkersX = 64;
template<typename T>
__global__ void bsplineXdirProcess(T *image, const size_t x_num, const size_t y_num, const size_t z_num,
                                   const float *bc1_vec, const float *bc2_vec, const float *bc3_vec, const float *bc4_vec,
                                   size_t k0, float b1, float b2, float norm_factor, float *boundary) {
    const int localId = threadIdx.x;
    const int yOffset = blockIdx.x * blockDim.x;
    const int64_t zOffset = blockIdx.z * blockDim.z * x_num * y_num;

    if (yOffset + localId < y_num) {
        float temp1 = 0;
        float temp2 = 0;
        float temp3 = 0;
        float temp4 = 0;
        // calculate boundary values
        for (int k = 0; k < k0; ++k) {
            T val = image[zOffset + k * y_num + yOffset + localId];
            temp1 += bc1_vec[k] * val;
            temp2 += bc2_vec[k] * val;
            val = image[zOffset + (x_num - 1 - k) * y_num + yOffset + localId];
            temp3 += bc3_vec[k] * val;
            temp4 += bc4_vec[k] * val;
        }
        // s
        image[zOffset + 0 * y_num + yOffset + localId] = temp2;
        image[zOffset + 1 * y_num + yOffset + localId] = temp1;
        image[zOffset + (x_num - 2) * y_num + yOffset + localId] = temp3 * norm_factor;
        image[zOffset + (x_num - 1) * y_num + yOffset + localId] = temp4 * norm_factor;

        int64_t offset = zOffset + 2 * y_num + yOffset + localId;
        int64_t offsetLimit = zOffset + (x_num - 2) * y_num;
        do {
            // do calculations and store
            const float temp = temp1 * b1 + temp2 * b2 + image[offset];
            image[offset] = temp;
            temp2 = temp1;
            temp1 = temp;

            offset += y_num;
        } while (offset < offsetLimit);

        offset = zOffset + (x_num - 3) * y_num + yOffset + localId;
        offsetLimit = zOffset;
        do {
            // do calculations and store
            const float temp = temp3 * b1 + temp4 * b2 + image[offset];
            image[offset] = temp * norm_factor;
            temp4 = temp3;
            temp3 = temp;

            offset -= y_num;
        } while (offset >= offsetLimit);
    }
}

template <typename ImgType>
void cudaFilterBsplineXdirection(MeshData<ImgType> &input, float lambda, float tolerance) {
    APRTimer timer;
    timer.verbose_flag=true;

    BsplineParams p = prepareBsplineStuff(input, lambda, tolerance);

    timer.start_timer("cuda: memory alloc + data transfer to device");
    size_t inputSize = input.mesh.size() * sizeof(ImgType);
    ImgType *cudaInput;
    hipMalloc(&cudaInput, inputSize);
    hipMemcpy(cudaInput, input.mesh.get(), inputSize, hipMemcpyHostToDevice);
    float *boundary;
    int boundaryLen = sizeof(float) * 4 * input.x_num * input.z_num;
    hipMalloc(&boundary, boundaryLen);

    thrust::device_vector<float> d_bc1_vec(p.bc1_vec);
    thrust::device_vector<float> d_bc2_vec(p.bc2_vec);
    thrust::device_vector<float> d_bc3_vec(p.bc3_vec);
    thrust::device_vector<float> d_bc4_vec(p.bc4_vec);
    timer.stop_timer();

    hipDeviceSetCacheConfig(hipFuncCachePreferShared);
    hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);

    dim3 threadsPerBlock(numOfWorkersX, 1, 1);
    dim3 numBlocks((input.y_num + threadsPerBlock.x - 1)/threadsPerBlock.x,
                   1, //fixme
                   (input.z_num + threadsPerBlock.z - 1)/threadsPerBlock.z);
    std::cout << "Number of blocks  (x/y/z):  " << numBlocks.x << "/" << numBlocks.y << "/" << numBlocks.z << std::endl;
    std::cout << "Number of threads (x/y/z): " << threadsPerBlock.x << "/" << threadsPerBlock.y << "/" << threadsPerBlock.z << std::endl;

    float *bc1 = thrust::raw_pointer_cast(d_bc1_vec.data());
    float *bc2 = thrust::raw_pointer_cast(d_bc2_vec.data());
    float *bc3 = thrust::raw_pointer_cast(d_bc3_vec.data());
    float *bc4 = thrust::raw_pointer_cast(d_bc4_vec.data());

    timer.start_timer("cuda: calculations on device ============================================================================ ");
        bsplineXdirProcess<ImgType> <<< numBlocks, threadsPerBlock >>>
                (cudaInput, input.x_num, input.y_num, input.z_num, bc1, bc2, bc3, bc4, p.k0, p.b1, p.b2, p.norm_factor, boundary);

    hipDeviceSynchronize();

    timer.stop_timer();

    timer.start_timer("cuda: transfer data from device and freeing memory");
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)printf("Error: %s\n", hipGetErrorString(err));

    hipMemcpy((void*)input.mesh.get(), cudaInput, inputSize, hipMemcpyDeviceToHost);
    hipFree(cudaInput);
    timer.stop_timer();
}

template<typename T>
__global__ void bsplineZdirProcess(T *image, const size_t x_num, const size_t y_num, const size_t z_num,
                                   const float *bc1_vec, const float *bc2_vec, const float *bc3_vec, const float *bc4_vec,
                                   size_t k0, float b1, float b2, float norm_factor, float *boundary) {
    const int localId = threadIdx.x;
    const int yOffset = blockIdx.x * blockDim.x;
    const int64_t xOffset = blockIdx.z * blockDim.z * y_num;
    const int64_t zDirOffset = x_num * y_num;

    if (yOffset + localId < y_num) {
        float temp1 = 0;
        float temp2 = 0;
        float temp3 = 0;
        float temp4 = 0;
        // calculate boundary values
        for (int k = 0; k < k0; ++k) {
            T val = image[xOffset + k * zDirOffset + yOffset + localId];
            temp1 += bc1_vec[k] * val;
            temp2 += bc2_vec[k] * val;
            val = image[xOffset + (z_num - 1 - k) * zDirOffset + yOffset + localId];
            temp3 += bc3_vec[k] * val;
            temp4 += bc4_vec[k] * val;
        }
        // s
        image[xOffset + 0 * zDirOffset + yOffset + localId] = temp2;
        image[xOffset + 1 * zDirOffset  + yOffset + localId] = temp1;
        image[xOffset + (z_num - 2) * zDirOffset + yOffset + localId] = temp3 * norm_factor;
        image[xOffset + (z_num - 1) * zDirOffset + yOffset + localId] = temp4 * norm_factor;

        int64_t offset = xOffset + 2 * zDirOffset + yOffset + localId;
        int64_t offsetLimit = xOffset + (z_num - 2) * zDirOffset;
        do {
            // do calculations and store
            const float temp = temp1 * b1 + temp2 * b2 + image[offset];
            image[offset] = temp;
            temp2 = temp1;
            temp1 = temp;

            offset += zDirOffset;
        } while (offset < offsetLimit);

        offset = xOffset + (z_num - 3) * zDirOffset + yOffset + localId;
        offsetLimit = xOffset;
        do {
            // do calculations and store
            const float temp = temp3 * b1 + temp4 * b2 + image[offset];
            image[offset] = temp * norm_factor;
            temp4 = temp3;
            temp3 = temp;

            offset -= zDirOffset;
        } while (offset >= offsetLimit);
    }
}

template <typename ImgType>
void cudaFilterBsplineZdirection(MeshData<ImgType> &input, float lambda, float tolerance) {
    APRTimer timer;
    timer.verbose_flag=true;

    BsplineParams p = prepareBsplineStuff(input, lambda, tolerance);

    timer.start_timer("cuda: memory alloc + data transfer to device");
    size_t inputSize = input.mesh.size() * sizeof(ImgType);
    ImgType *cudaInput;
    hipMalloc(&cudaInput, inputSize);
    hipMemcpy(cudaInput, input.mesh.get(), inputSize, hipMemcpyHostToDevice);
    float *boundary;
    int boundaryLen = sizeof(float) * 4 * input.x_num * input.z_num;
    hipMalloc(&boundary, boundaryLen);

    thrust::device_vector<float> d_bc1_vec(p.bc1_vec);
    thrust::device_vector<float> d_bc2_vec(p.bc2_vec);
    thrust::device_vector<float> d_bc3_vec(p.bc3_vec);
    thrust::device_vector<float> d_bc4_vec(p.bc4_vec);
    timer.stop_timer();

    hipDeviceSetCacheConfig(hipFuncCachePreferShared);
    hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);

    dim3 threadsPerBlock(numOfWorkersX, 1, 1);
    dim3 numBlocks((input.y_num + threadsPerBlock.x - 1)/threadsPerBlock.x,
                   1, //fixme
                   (input.x_num + threadsPerBlock.z - 1)/threadsPerBlock.z);
    std::cout << "Number of blocks  (x/y/z):  " << numBlocks.x << "/" << numBlocks.y << "/" << numBlocks.z << std::endl;
    std::cout << "Number of threads (x/y/z): " << threadsPerBlock.x << "/" << threadsPerBlock.y << "/" << threadsPerBlock.z << std::endl;

    float *bc1 = thrust::raw_pointer_cast(d_bc1_vec.data());
    float *bc2 = thrust::raw_pointer_cast(d_bc2_vec.data());
    float *bc3 = thrust::raw_pointer_cast(d_bc3_vec.data());
    float *bc4 = thrust::raw_pointer_cast(d_bc4_vec.data());

    timer.start_timer("cuda: calculations on device ============================================================================ ");
    bsplineZdirProcess<ImgType> <<< numBlocks, threadsPerBlock >>>
                                               (cudaInput, input.x_num, input.y_num, input.z_num, bc1, bc2, bc3, bc4, p.k0, p.b1, p.b2, p.norm_factor, boundary);

    hipDeviceSynchronize();

    timer.stop_timer();

    timer.start_timer("cuda: transfer data from device and freeing memory");
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)printf("Error: %s\n", hipGetErrorString(err));

    hipMemcpy((void*)input.mesh.get(), cudaInput, inputSize, hipMemcpyDeviceToHost);
    hipFree(cudaInput);
    timer.stop_timer();
}

void emptyCallForTemplateInstantiation() {
    MeshData<float> f = MeshData<float>(0,0,0);
    MeshData<uint16_t> u16 = MeshData<uint16_t>(0,0,0);

    cudaFilterBsplineYdirection(f, 3, 0.1);
    cudaFilterBsplineYdirection(u16, 3, 0.1);

    cudaFilterBsplineXdirection(f, 3, 0.1);
    cudaFilterBsplineXdirection(u16, 3, 0.1);

    cudaFilterBsplineZdirection(f, 3, 0.1);
    cudaFilterBsplineZdirection(u16, 3, 0.1);
}