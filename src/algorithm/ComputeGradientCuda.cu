#include "hip/hip_runtime.h"
#include "ComputeGradientCuda.hpp"
#include <iostream>
#include <memory>

#include <hip/hip_runtime.h>
#include <>


__global__ void gradient(float *input, size_t x_num, size_t y_num, size_t z_num, float *grad, size_t x_num_ds, size_t y_num_ds, float hx, float hy, float hz) {
    const int xi = ((blockIdx.x * blockDim.x) + threadIdx.x) * 2;
    const int yi = ((blockIdx.y * blockDim.y) + threadIdx.y) * 2;
    const int zi = ((blockIdx.z * blockDim.z) + threadIdx.z) * 2;
    if (xi >= x_num || yi >= y_num || zi >= z_num) return;

    const size_t xnumynum = x_num * y_num;

    float temp[4][4][4];

    for (int z = 0; z < 4; ++z)
        for (int x = 0; x < 4; ++x)
            for(int y = 0; y < 4; ++y) {
                int xc = xi + x - 1; if (xc < 0) xc = 0; else if (xc >= x_num) xc = x_num - 1;
                int yc = yi + y - 1; if (yc < 0) yc = 0; else if (yc >= y_num) yc = y_num - 1;
                int zc = zi + z - 1; if (zc < 0) zc = 0; else if (zc >= z_num) zc = z_num - 1;
                temp[z][x][y] = *(input + zc * xnumynum + xc * y_num + yc);
            }
    float maxGrad = 0;
    for (int z = 1; z <= 2; ++z)
        for (int x = 1; x <= 2; ++x)
            for(int y = 1; y <= 2; ++y) {
                float xd = (temp[z][x-1][y] - temp[z][x+1][y]) / (2 * hx); xd = xd * xd;
                float yd = (temp[z-1][x][y] - temp[z+1][x][y]) / (2 * hy); yd = yd * yd;
                float zd = (temp[z][x][y-1] - temp[z][x][y+1]) / (2 * hz); zd = zd * zd;
                float gm = __fsqrt_rn(xd + yd + zd);
                if (gm > maxGrad)  maxGrad = gm;
            }

    const size_t idx = zi/2 * x_num_ds * y_num_ds + xi/2 * y_num_ds + yi/2;
    grad[idx] = maxGrad;
}

void cudaDownsampledGradient(const MeshData<float> &input, MeshData<float> &grad, const float hx, const float hy,const float hz) {
    APRTimer timer;
    timer.verbose_flag=true;

    timer.start_timer("cuda: memory alloc + data transfer to device");
    size_t inputSize = input.mesh.size() * sizeof(float);
    float *cudaInput;
    hipMalloc(&cudaInput, inputSize);
    hipMemcpy(cudaInput, input.mesh.get(), inputSize, hipMemcpyHostToDevice);

    size_t gradSize = grad.mesh.size() * sizeof(float);
    float *cudaGrad;
    hipMalloc(&cudaGrad, gradSize);
    timer.stop_timer();

    timer.start_timer("cuda: calculations on device");
    dim3 threadsPerBlock(1, 32, 1);
    dim3 numBlocks((input.x_num + threadsPerBlock.x - 1)/threadsPerBlock.x,
                   (input.y_num + threadsPerBlock.y - 1)/threadsPerBlock.y,
                   (input.z_num + threadsPerBlock.z - 1)/threadsPerBlock.z);
    std::cout << "Number of blocks  (x/y/z):  " << numBlocks.x << "/" << numBlocks.y << "/" << numBlocks.z << std::endl;
    std::cout << "Number of threads (x/y/z): " << threadsPerBlock.x << "/" << threadsPerBlock.y << "/" << threadsPerBlock.z << std::endl;

    gradient<<<numBlocks,threadsPerBlock>>>(cudaInput, input.x_num, input.y_num, input.z_num, cudaGrad, grad.x_num, grad.y_num, hx, hy, hz);
    hipDeviceSynchronize();
    timer.stop_timer();

    timer.start_timer("cuda: transfer data from device and freeing memory");
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)printf("Error: %s\n", hipGetErrorString(err));
    hipMemcpy((void*)input.mesh.get(), cudaInput, inputSize, hipMemcpyDeviceToHost);
    hipFree(cudaInput);
    hipMemcpy((void*)grad.mesh.get(), cudaGrad, gradSize, hipMemcpyDeviceToHost);
    hipFree(cudaGrad);
    timer.stop_timer();
}
