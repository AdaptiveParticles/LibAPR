#include "hip/hip_runtime.h"
#include "ComputeBsplineRecursiveFilterCuda.h"
#include <iostream>
#include <memory>

#include <hip/hip_runtime.h>
#include <>

#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>

#include "bsplineXdir.cuh"
#include "bsplineYdir.cuh"
#include "bsplineZdir.cuh"
#include "misc/CudaTools.hpp"


namespace {
    typedef struct {
        std::vector<float> bc1;
        std::vector<float> bc2;
        std::vector<float> bc3;
        std::vector<float> bc4;
        size_t k0;
        float b1;
        float b2;
        float norm_factor;
    } BsplineParams;

    float impulse_resp(float k, float rho, float omg) {
        //  Impulse Response Function
        return (pow(rho, (std::abs(k))) * sin((std::abs(k) + 1) * omg)) / sin(omg);
    }

    float impulse_resp_back(float k, float rho, float omg, float gamma, float c0) {
        //  Impulse Response Function (nominator eq. 4.8, denominator from eq. 4.7)
        return c0 * pow(rho, std::abs(k)) * (cos(omg * std::abs(k)) + gamma * sin(omg * std::abs(k))) *
               (1.0 / (pow((1 - 2.0 * rho * cos(omg) + pow(rho, 2)), 2)));
    }

    template<typename T>
    BsplineParams prepareBsplineStuff(PixelData<T> &image, float lambda, float tol, int k0Len = -1) {
        // Recursive Filter Implimentation for Smoothing BSplines
        // B-Spline Signal Processing: Part II - Efficient Design and Applications, Unser 1993

        float xi = 1 - 96 * lambda + 24 * lambda * sqrt(3 + 144 * lambda); // eq 4.6
        float rho = (24 * lambda - 1 - sqrt(xi)) / (24 * lambda) *
                    sqrt((1 / xi) * (48 * lambda + 24 * lambda * sqrt(3 + 144 * lambda))); // eq 4.5
        float omg = atan(sqrt((1 / xi) * (144 * lambda - 1))); // eq 4.6

        float c0 = (1 + pow(rho, 2)) / (1 - pow(rho, 2)) * (1 - 2 * rho * cos(omg) + pow(rho, 2)) /
                   (1 + 2 * rho * cos(omg) + pow(rho, 2)); // eq 4.8
        float gamma = (1 - pow(rho, 2)) / (1 + pow(rho, 2)) * (1 / tan(omg)); // eq 4.8

        const float b1 = 2 * rho * cos(omg);
        const float b2 = -pow(rho, 2.0);

        const size_t idealK0Len = ceil(std::abs(log(tol) / log(rho)));
        const size_t minDimension = std::min(image.z_num, std::min(image.x_num, image.y_num));
        const size_t k0 = k0Len > 0 ? k0Len : std::min(idealK0Len, minDimension);

        const float norm_factor = pow((1 - 2.0 * rho * cos(omg) + pow(rho, 2)), 2);
        std::cout << "GPU: xi=" << xi << " rho=" << rho << " omg=" << omg << " gamma=" << gamma << " b1=" << b1
                  << " b2=" << b2 << " k0=" << k0 << " norm_factor=" << norm_factor << std::endl;

        // ------- Calculating boundary conditions

        // forward boundaries
        std::vector<float> impulse_resp_vec_f(k0 + 1);
        for (size_t k = 0; k < impulse_resp_vec_f.size(); ++k) impulse_resp_vec_f[k] = impulse_resp(k, rho, omg);

        //y(0) init
        std::vector<float> bc1(k0, 0);
        for (size_t k = 0; k < k0; ++k) bc1[k] = impulse_resp_vec_f[k];
        //y(1) init
        std::vector<float> bc2(k0, 0);
        bc2[1] = impulse_resp_vec_f[0];
        for (size_t k = 0; k < k0; ++k) bc2[k] += impulse_resp_vec_f[k + 1];

        // backward boundaries
        std::vector<float> impulse_resp_vec_b(k0 + 1);
        for (size_t k = 0; k < impulse_resp_vec_b.size(); ++k)
            impulse_resp_vec_b[k] = impulse_resp_back(k, rho, omg, gamma, c0);

        //y(N-1) init
        std::vector<float> bc3(k0, 0);
        bc3[0] = impulse_resp_vec_b[1];
        for (size_t k = 0; k < (k0 - 1); ++k) bc3[k + 1] += impulse_resp_vec_b[k] + impulse_resp_vec_b[k + 2];
        //y(N) init
        std::vector<float> bc4(k0, 0);
        bc4[0] = impulse_resp_vec_b[0];
        for (size_t k = 1; k < k0; ++k) bc4[k] += 2 * impulse_resp_vec_b[k];


        return BsplineParams{
                bc1,
                bc2,
                bc3,
                bc4,
                k0,
                b1,
                b2,
                norm_factor
        };
    }
}

// explicit instantiation of handled types
template void cudaFilterBsplineFull(PixelData<float> &, float, float, TypeOfRecBsplineFlags, int);


template <typename ImgType>
void cudaFilterBsplineFull(PixelData<ImgType> &input, float lambda, float tolerance, TypeOfRecBsplineFlags flags, int k0Len) {
    APRTimer timer(true), timerFullPipelilne(true);
    size_t inputSize = input.mesh.size() * sizeof(ImgType);
    BsplineParams p = prepareBsplineStuff(input, lambda, tolerance, k0Len);

    timer.start_timer("GpuMemTransferHostToDevice");
    thrust::device_vector<float> d_bc1(p.bc1);
    thrust::device_vector<float> d_bc2(p.bc2);
    thrust::device_vector<float> d_bc3(p.bc3);
    thrust::device_vector<float> d_bc4(p.bc4);
    float *bc1= raw_pointer_cast(d_bc1.data());
    float *bc2= raw_pointer_cast(d_bc2.data());
    float *bc3= raw_pointer_cast(d_bc3.data());
    float *bc4= raw_pointer_cast(d_bc4.data());
    ImgType *cudaInput;
    hipMalloc(&cudaInput, inputSize);
    hipMemcpy(cudaInput, input.mesh.get(), inputSize, hipMemcpyHostToDevice);
    float *boundary;
    if (flags & BSPLINE_Y_DIR) {
        int boundaryLen = sizeof(float) * (2 /*two first elements*/ + 2 /* two last elements */) * input.x_num * input.z_num;
        hipMalloc(&boundary, boundaryLen);
    }
    timer.stop_timer();

//    hipDeviceSetCacheConfig(hipFuncCachePreferShared);
//    hipDeviceSetSharedMemConfig(hipSharedMemBankSizeFourByte);

    timerFullPipelilne.start_timer("GpuDeviceTimeFull");
    if (flags & BSPLINE_Y_DIR) {
        timer.start_timer("GpuDeviceTimeYdir");
        dim3 threadsPerBlock(numOfThreads);
        dim3 numBlocks((input.x_num * input.z_num + threadsPerBlock.x - 1) / threadsPerBlock.x);
        printCudaDims(threadsPerBlock, numBlocks);
        size_t sharedMemSize = (2 /*bc vectors*/) * (p.k0) * sizeof(float) + numOfThreads * (p.k0) * sizeof(ImgType);
        bsplineYdirBoundary<ImgType> <<< numBlocks, threadsPerBlock, sharedMemSize >>> (cudaInput, input.x_num, input.y_num, input.z_num, bc1, bc2, bc3, bc4, p.k0, boundary);
        sharedMemSize = numOfThreads * blockWidth * sizeof(ImgType);
        bsplineYdirProcess<ImgType> <<< numBlocks, threadsPerBlock, sharedMemSize >>> (cudaInput, input.x_num, input.y_num, input.z_num, p.k0, p.b1, p.b2, p.norm_factor, boundary);
        waitForCuda();
        hipFree(boundary);
        timer.stop_timer();
    }
    constexpr int numOfWorkersYdir = 64;
    if (flags & BSPLINE_X_DIR) {
        dim3 threadsPerBlockX(1, numOfWorkersYdir, 1);
        dim3 numBlocksX(1,
                        (input.y_num + threadsPerBlockX.y - 1) / threadsPerBlockX.y,
                        (input.z_num + threadsPerBlockX.z - 1) / threadsPerBlockX.z);
        printCudaDims(threadsPerBlockX, numBlocksX);
        timer.start_timer("GpuDeviceTimeXdir");
        bsplineXdir<ImgType> <<< numBlocksX, threadsPerBlockX >>> (cudaInput, input.x_num, input.y_num, bc1, bc2, bc3, bc4, p.k0, p.b1, p.b2, p.norm_factor);
        waitForCuda();
        timer.stop_timer();
    }
    if (flags & BSPLINE_Z_DIR) {
        dim3 threadsPerBlockZ(1, numOfWorkersYdir, 1);
        dim3 numBlocksZ(1,
                        (input.y_num + threadsPerBlockZ.y - 1) / threadsPerBlockZ.y,
                        (input.x_num + threadsPerBlockZ.x - 1) / threadsPerBlockZ.x); // Intentionally x-dim is here (after y) to get good memory coalescing
        printCudaDims(threadsPerBlockZ, numBlocksZ);
        timer.start_timer("GpuDeviceTimeZdir");
        bsplineZdir<ImgType> <<< numBlocksZ, threadsPerBlockZ >>> (cudaInput, input.x_num, input.y_num, input.z_num, bc1, bc2, bc3, bc4, p.k0, p.b1, p.b2, p.norm_factor);
        waitForCuda();
        timer.stop_timer();
    }
    timerFullPipelilne.stop_timer();

    timer.start_timer("GpuMemTransferDeviceToHost");
    getDataFromKernel(input, inputSize, cudaInput);
    timer.stop_timer();
}
