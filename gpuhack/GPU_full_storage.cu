#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>
#include <array>
#include <iostream>
#include <cassert>
#include <limits>

#include "data_structures/APR/APR.hpp"
#include "data_structures/APR/APRTreeIterator.hpp"
#include "data_structures/APR/ExtraParticleData.hpp"

#include "thrust/device_vector.h"
#include "thrust/tuple.h"
#include "thrust/copy.h"

struct cmdLineOptions{
    std::string output = "output";
    std::string stats = "";
    std::string directory = "";
    std::string input = "";
};

bool command_option_exists(char **begin, char **end, const std::string &option) {
    return std::find(begin, end, option) != end;
}

char* get_command_option(char **begin, char **end, const std::string &option) {
    char ** itr = std::find(begin, end, option);
    if (itr != end && ++itr != end) {
        return *itr;
    }
    return 0;
}

cmdLineOptions read_command_line_options(int argc, char **argv) {
    cmdLineOptions result;

    if(argc == 1) {
        std::cerr << "Usage: \"Example_apr_neighbour_access -i input_apr_file -d directory\"" << std::endl;
        exit(1);
    }
    if(command_option_exists(argv, argv + argc, "-i")) {
        result.input = std::string(get_command_option(argv, argv + argc, "-i"));
    } else {
        std::cout << "Input file required" << std::endl;
        exit(2);
    }
    if(command_option_exists(argv, argv + argc, "-d")) {
        result.directory = std::string(get_command_option(argv, argv + argc, "-d"));
    }
    if(command_option_exists(argv, argv + argc, "-o")) {
        result.output = std::string(get_command_option(argv, argv + argc, "-o"));
    }

    return result;
}

__global__ void copy_out(
    std::size_t _level,
    const thrust::tuple<std::size_t,std::size_t>* _line_offsets,
    const std::uint16_t*            _y_ex,
    const std::uint16_t*            _pdata,
    const std::size_t*            _offsets,
    std::uint16_t* _results,
    std::size_t max_x,
    std::size_t max_z,
    std::size_t nparticles
    ){

    unsigned int x_index = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int z_index = blockDim.y * blockIdx.y + threadIdx.y;

    if(x_index < max_x){
        return;
    }

    if(z_index < max_z){
        return;
    }

    auto level_zx_offset = _offsets[_level] + max_x * z_index + x_index;
    auto row_start = _line_offsets[level_zx_offset];

    if(thrust::get<1>(row_start) == 0)
        return;

    auto particle_index_begin = thrust::get<0>(row_start);
    auto particle_index_end   = thrust::get<1>(row_start);

    for (std::size_t global_index = particle_index_begin;
         global_index <= particle_index_end; ++global_index) {

        uint16_t current_particle_value = _pdata[global_index];

        _results[global_index] = current_particle_value;

    }


}
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char **argv) {
    // Read provided APR file
    cmdLineOptions options = read_command_line_options(argc, argv);
    std::string fileName = options.directory + options.input;
    APR<uint16_t> apr;
    apr.read_apr(fileName);

    // Get dense representation of APR
    APRIterator<uint16_t> aprIt(apr);

    ExtraParticleData<uint16_t> yRow;
    ExtraParticleData<uint16_t> xRow;
    ExtraParticleData<uint16_t> zRow;
    ExtraParticleData<uint16_t> levelRow;
    ExtraParticleData<uint64_t> globalIndexRow;


    uint64_t particle_number;
    //Basic serial iteration over all particles
    for (particle_number = 0; particle_number < aprIt.total_number_particles(); ++particle_number) {
        //This step is required for all loops to set the iterator by the particle number
        aprIt.set_iterator_to_particle_by_number(particle_number);

        //you can then also use it to access any particle properties stored as ExtraParticleData

        yRow.data.push_back(aprIt.y());
        xRow.data.push_back(aprIt.x());
        zRow.data.push_back(aprIt.z());
        levelRow.data.push_back(aprIt.level());
        globalIndexRow.data.push_back(aprIt.global_index());

    }


    ///////////////////////////
    ///
    /// Sparse Data for GPU
    ///
    ///////////////////////////

    std::vector<std::tuple<std::size_t,std::size_t>> level_zx_index_start;//size = number of rows on all levels
    std::vector<std::uint16_t> y_explicit;y_explicit.reserve(aprIt.total_number_particles());//size = number of particles
    std::vector<std::uint16_t> particle_values;particle_values.reserve(aprIt.total_number_particles());//size = number of particles
    std::vector<std::size_t> level_offset(aprIt.level_max()+1,UINT64_MAX);//size = number of levels

    std::size_t x = 0;
    std::size_t z = 0;

    std::size_t zx_counter = 0;


    for (int level = aprIt.level_min(); level <= aprIt.level_max(); ++level) {
        level_offset[level] = zx_counter;

        for (z = 0; z < aprIt.spatial_index_z_max(level); ++z) {
            for (x = 0; x < aprIt.spatial_index_x_max(level); ++x) {

                zx_counter++;
                if (aprIt.set_new_lzx(level, z, x) < UINT64_MAX) {
                    level_zx_index_start.emplace_back(std::make_tuple<std::size_t,std::size_t>(aprIt.global_index(),
                                                                                               aprIt.particles_zx_end(level,z,x)-1)); //This stores the begining and end global index for each level_xz_row
                } else {
                    level_zx_index_start.emplace_back(std::make_tuple<std::size_t,std::size_t>(UINT64_MAX, 0)); //This stores the begining and end global index for each level_
                }

                for (aprIt.set_new_lzx(level, z, x);
                     aprIt.global_index() < aprIt.particles_zx_end(level, z,
                                                                   x); aprIt.set_iterator_to_particle_next_particle()) {
                    y_explicit.emplace_back(aprIt.y());
                    particle_values.emplace_back(apr.particles_intensities[aprIt]);

                }
            }

        }
    }


    ////////////////////
    ///
    /// Example of doing our level,z,x access using the GPU data structure
    ///
    /////////////////////

    thrust::device_vector<std::uint16_t> d_test_access_data(apr.particles_intensities.data.size(),std::numeric_limits<std::uint16_t>::max());

    thrust::host_vector<thrust::tuple<std::size_t,std::size_t> > h_level_zx_index_start(level_zx_index_start.size());
    thrust::transform(level_zx_index_start.begin(), level_zx_index_start.end(),
                      h_level_zx_index_start.begin(),
                      [] ( const auto& _el ){
                          return thrust::make_tuple(std::get<0>(_el), std::get<1>(_el));
                      } );

    thrust::device_vector<thrust::tuple<std::size_t,std::size_t> > d_level_zx_index_start = h_level_zx_index_start;

    thrust::device_vector<std::uint16_t> d_y_explicit(y_explicit.begin(), y_explicit.end());
    thrust::device_vector<std::uint16_t> d_particle_values(particle_values.begin(), particle_values.end());
    thrust::device_vector<std::size_t> d_level_offset(level_offset.begin(),level_offset.end());

    const thrust::tuple<std::size_t,std::size_t>* levels =  thrust::raw_pointer_cast(d_level_zx_index_start.data());
    const std::uint16_t*             y_ex   =  thrust::raw_pointer_cast(d_y_explicit.data());
    const std::uint16_t*             pdata  =  thrust::raw_pointer_cast(d_particle_values.data());
    const std::size_t*             offsets= thrust::raw_pointer_cast(d_level_offset.data());
    std::uint16_t*                   result = thrust::raw_pointer_cast(d_test_access_data.data());


    for (int lvl = aprIt.level_min(); lvl <= aprIt.level_max(); ++lvl) {

        const int x_num = aprIt.spatial_index_x_max(lvl);
        const int z_num = aprIt.spatial_index_z_max(lvl);

        dim3 threads(32,4,1);
        dim3 blocks((x_num + threads.x- 1)/threads.x,
                    (z_num + threads.y- 1)/threads.y ,
                    1);

        copy_out<<<blocks,threads>>>(lvl,
                                     levels,
                                     y_ex,
                                     pdata,
                                     offsets,
                                     result,
                                     x_num,z_num,
                                     particle_values.size());


    }

    hipDeviceSynchronize();

    std::vector<std::uint16_t> test_access_data(d_test_access_data.size());
    thrust::copy(d_test_access_data.begin(), d_test_access_data.end(), test_access_data.begin());

    assert(test_access_data.front() != std::numeric_limits<std::uint16_t>::max());
    assert(test_access_data[0] != std::numeric_limits<std::uint16_t>::max());
    assert(test_access_data.back() != std::numeric_limits<std::uint16_t>::max());



    //////////////////////////
    ///
    /// Now check the data
    ///
    ////////////////////////////

    bool success = true;

    for (std::size_t i = 0; i < test_access_data.size(); ++i) {
        if(apr.particles_intensities.data[i]!=test_access_data[i]){
            success = false;
            std::cout << i << " expected: " << apr.particles_intensities.data[i] << ", received: " << test_access_data[i] << "\n";
            break;
        }
    }

    if(success){
        std::cout << "PASS" << std::endl;
    } else {
        std::cout << "FAIL" << std::endl;
    }


}
